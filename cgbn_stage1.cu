#include "hip/hip_runtime.h"
/***

Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the "Software"),
to deal in the Software without restriction, including without limitation
the rights to use, copy, modify, merge, publish, distribute, sublicense,
and/or sell copies of the Software, and to permit persons to whom the
Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
IN THE SOFTWARE.

***/

#include "cgbn_stage1.h"

#include <cassert>
#include <stdio.h>
#include <stdlib.h>

// GMP import must proceed cgbn.h
#include <gmp.h>
#include <cgbn.h>
#include <hip/hip_runtime.h>

#include "ecm.h"
#include "ecm-gpu.h"


void cuda_check(hipError_t status, const char *action=NULL, const char *file=NULL, int32_t line=0) {
  // check for cuda errors
  if (status!=hipSuccess) {
    fprintf (stderr, "CUDA error occurred: %s\n", hipGetErrorString(status));
    if (action!=NULL)
      fprintf (stderr, "While running %s   (file %s, line %d)\n", action, file, line);
    exit(1);
  }
}

// support routine copied from  "CGBN/samples/utility/support.h"
void cgbn_check(cgbn_error_report_t *report, const char *file=NULL, int32_t line=0) {
  // check for cgbn errors

  if(cgbn_error_report_check(report)) {
    fprintf (stderr, "\n");
    fprintf (stderr, "CGBN error occurred: %s\n", cgbn_error_string(report));

    if(report->_instance!=0xFFFFFFFF) {
      fprintf (stderr, "Error reported by instance %d", report->_instance);
      if(report->_blockIdx.x!=0xFFFFFFFF || report->_threadIdx.x!=0xFFFFFFFF)
        fprintf (stderr, ", ");
      if(report->_blockIdx.x!=0xFFFFFFFF)
      fprintf (stderr, "blockIdx=(%d, %d, %d) ", report->_blockIdx.x, report->_blockIdx.y, report->_blockIdx.z);
      if(report->_threadIdx.x!=0xFFFFFFFF)
        fprintf (stderr, "threadIdx=(%d, %d, %d)", report->_threadIdx.x, report->_threadIdx.y, report->_threadIdx.z);
      fprintf (stderr, "\n");
    }
    else {
      fprintf (stderr, "Error reported by blockIdx=(%d %d %d)", report->_blockIdx.x, report->_blockIdx.y, report->_blockIdx.z);
      fprintf (stderr, "threadIdx=(%d %d %d)\n", report->_threadIdx.x, report->_threadIdx.y, report->_threadIdx.z);
    }
    if(file!=NULL)
      fprintf (stderr, "file %s, line %d\n", file, line);
    exit(1);
  }
}

// Unify this with cudakernel.cu
#define CUDA_CHECK(action) cuda_check(action, #action, __FILE__, __LINE__)
#define CGBN_CHECK(report) cgbn_check(report, __FILE__, __LINE__)

static
void to_mpz(mpz_t r, const uint32_t *x, uint32_t count) {
  mpz_import(r, count, -1, sizeof(uint32_t), 0, 0, x);
}

static
void from_mpz(const mpz_t s, uint32_t *x, uint32_t count) {
  size_t words;

  if(mpz_sizeinbase(s, 2)>count*32) {
    fprintf (stderr, "from_mpz failed -- result does not fit\n");
    exit(1);
  }

  mpz_export(x, &words, -1, sizeof(uint32_t), 0, 0, s);
  while(words<count)
    x[words++]=0;
}



// ---------------------------------------------------------------- //

// See cgbn_error_t enum (cgbn.h:39)
#define cgbn_normalized_error ((cgbn_error_t) 14)
#define cgbn_positive_overflow ((cgbn_error_t) 15)
#define cgbn_negative_overflow ((cgbn_error_t) 16)

// Seems to adds very small overhead (1-10%)
#define VERIFY_NORMALIZED 1
// Adds even less overhead (<1%)
#define CHECK_ERROR 1

// Can dramatically change compile time
#if 1
    #define FORCE_INLINE __forceinline__
#else
    #define FORCE_INLINE
#endif


// The CGBN context uses the following three parameters:
//   TBP             - threads per block (zero means to use the blockDim.x)
//   MAX_ROTATION    - must be small power of 2, imperically, 4 works well
//   CONSTANT_TIME   - require constant time algorithms (currently, constant time algorithms are not available)

// Locally it will also be helpful to have several parameters:
//   TPI             - threads per instance
//   BITS            - number of bits per instance

template<uint32_t tpi, uint32_t bits>
class cgbn_params_t {
  public:
  // parameters used by the CGBN context
  static const uint32_t TPB=128;                   // Reasonable default
  static const uint32_t MAX_ROTATION=4;            // good default value
  static const uint32_t SHM_LIMIT=0;               // no shared mem available
  static const bool     CONSTANT_TIME=false;       // not implemented

  // parameters used locally in the application
  static const uint32_t TPI=tpi;                   // threads per instance
  static const uint32_t BITS=bits;                 // instance size
};


template<class params>
class curve_t {
  public:

  typedef cgbn_context_t<params::TPI, params>   context_t;
  typedef cgbn_env_t<context_t, params::BITS>   env_t;
  typedef typename env_t::cgbn_t                bn_t;
  typedef cgbn_mem_t<params::BITS>              mem_t;

  context_t _context;
  env_t     _env;
  int32_t   _instance; // which curve instance is this

  // Constructor
  __device__ FORCE_INLINE curve_t(cgbn_monitor_t monitor, cgbn_error_report_t *report, int32_t instance) :
      _context(monitor, report, (uint32_t)instance), _env(_context), _instance(instance) {}

  // Verify 0 <= r < modulus
  __device__ FORCE_INLINE void assert_normalized(bn_t &r, const bn_t &modulus) {
    //if (VERIFY_NORMALIZED && _context.check_errors())
    if (VERIFY_NORMALIZED && CHECK_ERROR) {

        // Negative overflow
        if (cgbn_extract_bits_ui32(_env, r, params::BITS-1, 1)) {
            _context.report_error(cgbn_negative_overflow);
        }
        // Positive overflow
        if (cgbn_compare(_env, r, modulus) >= 0) {
            _context.report_error(cgbn_positive_overflow);
        }
    }
  }

  // Normalize after addition
  __device__ FORCE_INLINE void normalize_addition(bn_t &r, const bn_t &modulus) {
      if (cgbn_compare(_env, r, modulus) >= 0) {
          cgbn_sub(_env, r, r, modulus);
      }
  }

  // Normalize after subtraction (handled instead by checking carry)
  /*
  __device__ FORCE_INLINE void normalize_subtraction(bn_t &r, const bn_t &modulus) {
      if (cgbn_extract_bits_ui32(_env, r, params::BITS-1, 1)) {
          cgbn_add(_env, r, r, modulus);
      }
  }
  */

  /**
   * Calculate (r * m) / 2^32 mod modulus
   *
   * This removes a factor of 2^32 which is not present in m.
   * Otherwise m (really d) needs to be passed as a bigint not a uint32
   */
  __device__ FORCE_INLINE void special_mult_ui32(bn_t &r, uint32_t m, const bn_t &modulus, uint32_t np0) {
    //uint32_t thread_i = (blockIdx.x*blockDim.x + threadIdx.x)%params::TPI;
    bn_t temp;

    uint32_t carry_t1 = cgbn_mul_ui32(_env, r, r, m);
    uint32_t t1_0 = cgbn_extract_bits_ui32(_env, r, 0, 32);
    uint32_t q = t1_0 * np0;
    uint32_t carry_t2 = cgbn_mul_ui32(_env, temp, modulus, q);

    cgbn_shift_right(_env, r, r, 32);
    cgbn_shift_right(_env, temp, temp, 32);
    // Add back overflow carry
    cgbn_insert_bits_ui32(_env, r, r, params::BITS-32, 32, carry_t1);
    cgbn_insert_bits_ui32(_env, temp, temp, params::BITS-32, 32, carry_t2);

    // This needs to be measured at block containing top bit of modulus
    int32_t carry_q = cgbn_add(_env, r, r, temp);
    carry_q += cgbn_add_ui32(_env, r, r, t1_0 != 0); // add 1
    while (carry_q != 0) {
        carry_q -= cgbn_sub(_env, r, r, modulus);
    }

    // 0 <= r, temp < modulus => r + temp + 1 < 2*modulus
    if (cgbn_compare(_env, r, modulus) >= 0) {
        cgbn_sub(_env, r, r, modulus);
    }
  }


  __device__ FORCE_INLINE void double_add_v2(
          bn_t &q, bn_t &u,
          bn_t &w, bn_t &v,
          uint32_t d,
          uint32_t bit_number,
          const bn_t &modulus,
          const uint32_t np0) {
    // q = xA = aX
    // u = zA = aY
    // w = xB = bX
    // v = zB = bY

    /* Doesn't seem to be a large cost to using many extra variables */
    bn_t t, CB, DA, AA, BB, K, dK;

    /* Can maybe use one more bit if cgbn_add subtracts when carry happens */

    cgbn_add(_env, t, v, w); // t = (bY + bX)
    normalize_addition(t, modulus);
    if (cgbn_sub(_env, v, v, w)) // v = (bY - bX)
        cgbn_add(_env, v, v, modulus);


    cgbn_add(_env, w, u, q); // w = (aY + aX)
    normalize_addition(w, modulus);
    if (cgbn_sub(_env, u, u, q)) // u = (aY - aX)
        cgbn_add(_env, u, u, modulus);
    if (VERIFY_NORMALIZED) {
        assert_normalized(t, modulus);
        assert_normalized(v, modulus);
        assert_normalized(w, modulus);
        assert_normalized(u, modulus);
    }

    cgbn_mont_mul(_env, CB, t, u, modulus, np0); // C*B
        normalize_addition(CB, modulus);
    cgbn_mont_mul(_env, DA, v, w, modulus, np0); // D*A
        normalize_addition(DA, modulus);

    cgbn_mont_sqr(_env, AA, w, modulus, np0);    // AA
    cgbn_mont_sqr(_env, BB, u, modulus, np0);    // BB
    normalize_addition(AA, modulus);
    normalize_addition(BB, modulus);
    if (VERIFY_NORMALIZED) {
        assert_normalized(CB, modulus);
        assert_normalized(DA, modulus);
        assert_normalized(AA, modulus);
        assert_normalized(BB, modulus);
    }

    // q = aX is finalized
    cgbn_mont_mul(_env, q, AA, BB, modulus, np0); // AA*BB
        normalize_addition(q, modulus);
        assert_normalized(q, modulus);

    if (cgbn_sub(_env, K, AA, BB)) // K = AA-BB
        cgbn_add(_env, K, K, modulus);

    // By definition of d = (sigma / 2^32) % MODN
    // K = k*R
    // dK = d*k*R = (K * R * sigma) >> 32
    cgbn_set(_env, dK, K);
    special_mult_ui32(dK, d, modulus, np0); // dK = K*d
        assert_normalized(dK, modulus);

    cgbn_add(_env, u, BB, dK); // BB + dK
    normalize_addition(u, modulus);
    if (VERIFY_NORMALIZED) {
        assert_normalized(K, modulus);
        assert_normalized(dK, modulus);
        assert_normalized(u, modulus);
    }

    // u = aY is finalized
    cgbn_mont_mul(_env, u, K, u, modulus, np0); // K(BB+dK)
        normalize_addition(u, modulus);
        assert_normalized(u, modulus);

    cgbn_add(_env, w, DA, CB); // DA + CB
    normalize_addition(w, modulus);
    if (cgbn_sub(_env, v, DA, CB)) // DA - CB
        cgbn_add(_env, v, v, modulus);
    if (VERIFY_NORMALIZED) {
        assert_normalized(w, modulus);
        assert_normalized(v, modulus);
    }

    // w = bX is finalized
    cgbn_mont_sqr(_env, w, w, modulus, np0); // (DA+CB)^2 mod N
        normalize_addition(w, modulus);
        assert_normalized(w, modulus);

    cgbn_mont_sqr(_env, v, v, modulus, np0); // (DA-CB)^2 mod N
        normalize_addition(v, modulus);
        assert_normalized(v, modulus);

    // v = bY is finalized
    cgbn_shift_left(_env, v, v, 1); // double
    normalize_addition(v, modulus);
        assert_normalized(v, modulus);
  }

  __host__ static mem_t* set_p_2p(const mpz_t N, const mpz_t s, uint32_t curves, uint32_t sigma) {
    // P1_x, P1_y = (2,1)
    // 2P_x, 2P_y = (9, 64 * d + 8)

    /** Consider curve copies of N (AKA modulo) */
    mem_t *data = (mem_t*) malloc(sizeof(mem_t) * 5 * curves);
    mem_t *datum = data;

    mpz_t x;
    mpz_init(x);
    for(int index = 0; index < curves; index++) {

        // d = (sigma / 2^32) mod N BUT 2^32 handled by special_mul_ui32
        uint32_t d = sigma + index;

        // mod
        from_mpz(N, (datum++)->_limbs, params::BITS/32);

        // P1 (X, Y)
        mpz_set_ui(x, 2);
        from_mpz(x, (datum++)->_limbs, params::BITS/32);
        mpz_set_ui(x, 1);
        from_mpz(x, (datum++)->_limbs, params::BITS/32);

        // 2P = P2 (X, Y)
        // P2_y = 64 * d + 8
        mpz_set_ui(x, 9);
        from_mpz(x, (datum++)->_limbs, params::BITS/32);

        // d = sigma * mod_inverse(2 ** 32, N)
        mpz_ui_pow_ui(x, 2, 32);
        mpz_invert(x, x, N);
        mpz_mul_ui(x, x, d);
        // P2_y = 64 * d - 2;
        mpz_mul_ui(x, x, 64);
        mpz_add_ui(x, x, 8);
        mpz_mod(x, x, N);

        outputf (OUTPUT_TRACE, "sigma %d => P2_y: %Zd\n", d, x); 
        from_mpz(x, (datum++)->_limbs, params::BITS/32);
    }

    mpz_clear(x);
    return data;
  }
};

// kernel implementation using cgbn
template<class params>
__global__ void kernel_double_add(
        cgbn_error_report_t *report,
        uint32_t num_bits,
        char* gpu_s_bits,
        typename curve_t<params>::mem_t *data,
        uint32_t count,
        uint32_t sigma_0) {

  // decode an instance_i number from the blockIdx and threadIdx
  int32_t instance_i = (blockIdx.x*blockDim.x + threadIdx.x)/params::TPI;
  if(instance_i >= count)
    return;

  cgbn_monitor_t monitor = CHECK_ERROR ? cgbn_report_monitor : cgbn_no_checks;

  curve_t<params> curve(monitor, report, instance_i);
  typename curve_t<params>::bn_t  aX, aY, bX, bY, modulus;

  uint32_t np0;
  { // Setup
      cgbn_load(curve._env, modulus, &data[5*instance_i+0]);
      cgbn_load(curve._env, aX, &data[5*instance_i+1]);
      cgbn_load(curve._env, aY, &data[5*instance_i+2]);
      cgbn_load(curve._env, bX, &data[5*instance_i+3]);
      cgbn_load(curve._env, bY, &data[5*instance_i+4]);

      // Convert everything to mont
      np0 = cgbn_bn2mont(curve._env, aX, aX, modulus);
      cgbn_bn2mont(curve._env, aY, aY, modulus);
      cgbn_bn2mont(curve._env, bX, bX, modulus);
      cgbn_bn2mont(curve._env, bY, bY, modulus);

      {
        curve.assert_normalized(aX, modulus);
        curve.assert_normalized(aY, modulus);
        curve.assert_normalized(bX, modulus);
        curve.assert_normalized(bY, modulus);
      }
  }

  uint32_t d = sigma_0 + instance_i;

  for (int b = num_bits; b > 0; b--) {
    /**
     * TODO generates a lot of duplicate inlined code, not sure how to improve
     * Tried with swappings pointers (with a single call to double_add_v2)
     */
    if (gpu_s_bits[num_bits - b] == 0) {
        curve.double_add_v2(aX, aY, bX, bY, d, b, modulus, np0);
    } else {
        curve.double_add_v2(bX, bY, aX, aY, d, b, modulus, np0);
    }
  }

  { // Final output
    // Convert everything back to bn
    cgbn_mont2bn(curve._env, aX, aX, modulus, np0);
    cgbn_mont2bn(curve._env, aY, aY, modulus, np0);
    cgbn_mont2bn(curve._env, bX, bX, modulus, np0);
    cgbn_mont2bn(curve._env, bY, bY, modulus, np0);
    {
      curve.assert_normalized(aX, modulus);
      curve.assert_normalized(aY, modulus);
      curve.assert_normalized(bX, modulus);
      curve.assert_normalized(bY, modulus);
    }
    cgbn_store(curve._env, &data[5*instance_i+1], aX);
    cgbn_store(curve._env, &data[5*instance_i+2], aY);
    cgbn_store(curve._env, &data[5*instance_i+3], bX);
    cgbn_store(curve._env, &data[5*instance_i+4], bY);
  }
}

static
int findfactor(mpz_t factor, const mpz_t N, const mpz_t x_final, const mpz_t y_final) {
    // XXX: combine / refactor logic with cudawrapper.c findfactor

    mpz_t temp;
    mpz_init(temp);

    // Check if factor found

    bool inverted = mpz_invert(temp, y_final, N);    // aY ^ (N-2) % N
    if (inverted) {
        mpz_mul(temp, x_final, temp);         // aX * aY^-1
        mpz_mod(factor, temp, N);             // "Residual"
        mpz_clear(temp);
        return ECM_NO_FACTOR_FOUND;
    }
    mpz_clear(temp);

    mpz_gcd(factor, y_final, N);
    return ECM_FACTOR_FOUND_STEP1;
}


static
int verify_size_of_n(const mpz_t N, size_t max_bits) {
  size_t n_log2 = mpz_sizeinbase(N, 2);

  // using check_gpuecm.sage it looks like 4 bits would suffice
  size_t max_usable_bits = max_bits - 6;

  if (n_log2 <= max_usable_bits)
    return ECM_NO_FACTOR_FOUND;

  outputf (OUTPUT_ERROR, "GPU: N(%d bits) + carry(%d bits) > BITS(%d)\n",
      n_log2, max_bits - max_usable_bits, max_bits);
  outputf (OUTPUT_ERROR, "GPU: Error, input number should be stricly lower than 2^%d\n",
      max_usable_bits);
  return ECM_ERROR;
}


static
char* allocate_and_set_s_bits(const mpz_t s, int *nbits) {
  uint32_t num_bits = *nbits = mpz_sizeinbase(s, 2) - 1;
  assert( 1 <= num_bits <= 100'000'000 );

  num_bits = num_bits;
  // Use int* so that size can be stored in first element, could pass around extra size.
  char *s_bits = (char*) malloc(sizeof(char) * num_bits);
  for (int i = 0; i < num_bits; i++) {
      s_bits[i] = mpz_tstbit (s, num_bits - 1 - i);
  }
  return s_bits;
}

static
int process_results(mpz_t *factors, int *array_stage_found,
                    const mpz_t N,
                    const uint32_t *data, uint32_t cgbn_bits,
                    int curves, uint32_t sigma) {
  mpz_t x_final, y_final, modulo;
  mpz_init(modulo);
  mpz_init(x_final);
  mpz_init(y_final);

  const uint32_t limbs_per = cgbn_bits / 32;

  int youpi = ECM_NO_FACTOR_FOUND;
  for(size_t i = 0; i < curves; i++) {
    const uint32_t *datum = data + (5 * i * limbs_per);;

    if (test_verbose (OUTPUT_TRACE) && i == 0) {
      to_mpz(modulo, datum + 0 * limbs_per, limbs_per);
      outputf (OUTPUT_TRACE, "index: 0 modulo: %Zd\n", modulo);

      to_mpz(x_final, datum + 1 * limbs_per, limbs_per);
      to_mpz(y_final, datum + 2 * limbs_per, limbs_per);
      outputf (OUTPUT_TRACE, "index: 0 pA: (%Zd, %Zd)\n", x_final, y_final);

      to_mpz(x_final, datum + 3 * limbs_per, limbs_per);
      to_mpz(y_final, datum + 4 * limbs_per, limbs_per);
      outputf (OUTPUT_TRACE, "index: 0 pB: (%Zd, %Zd)\n", x_final, y_final);
    }

    // Make sure we were testing the right number.
    to_mpz(modulo, datum + 0 * limbs_per, limbs_per);
    assert(mpz_cmp(modulo, N) == 0);

    to_mpz(x_final, datum + 1 * limbs_per, limbs_per);
    to_mpz(y_final, datum + 2 * limbs_per, limbs_per);

    array_stage_found[i] = findfactor(factors[i], N, x_final, y_final);
    if (array_stage_found[i] != ECM_NO_FACTOR_FOUND) {
      youpi = array_stage_found[i];
      outputf (OUTPUT_NORMAL, "GPU: factor %Zd found in Step 1 with curve %ld (-sigma %d:%d)\n",
          factors[i], i, ECM_PARAM_BATCH_32BITS_D, sigma + i);
    }
  }
  
  mpz_init(modulo);
  mpz_clear(x_final);
  mpz_clear(y_final);

  return youpi;
}

int run_cgbn(mpz_t *factors, int *array_stage_found,
             const mpz_t N, const mpz_t s, float *gputime,
             ecm_params_t *ecm_params) {

  const size_t MAX_BITS = 1024;

  size_t curves = ecm_params->curves;  
  assert( ecm_params->sigma > 0 );
  assert( ((uint64_t) ecm_params->sigma + curves) <= 0xFFFFFFFF ); // no overflow

  /* Validate N's size */
  int youpi = verify_size_of_n(N, MAX_BITS);
  if (youpi != ECM_NO_FACTOR_FOUND) {
    return youpi;
  }

  int num_bits;
  char *s_bits = allocate_and_set_s_bits(s, &num_bits);
  assert( 1 <= num_bits <= 100'000'000 );
  assert( s_bits != NULL );

  // Keeps CPU from busy waiting during GPU execution.
  CUDA_CHECK(hipSetDeviceFlags (hipDeviceScheduleBlockingSync));
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate (&start));
  CUDA_CHECK(hipEventCreate (&stop));
  CUDA_CHECK(hipEventRecord (start, 0));

  // Copy s_bits
  char     *gpu_s_bits;
  CUDA_CHECK(hipMalloc((void **)&gpu_s_bits, sizeof(char) * num_bits));
  CUDA_CHECK(hipMemcpy(gpu_s_bits, s_bits, sizeof(char) * num_bits, hipMemcpyHostToDevice));


  cgbn_error_report_t *report;
  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK(cgbn_error_report_alloc(&report));


  /**
   * Smaller TPI (e.g. 8) is faster (TPI=4 seems worse than TPI=8).
   * Larger TPI (e.g. 32) for running a single curve (or large N).
   * TPI=8  is required for N > 512
   * TPI=16 is required for N > 2048
   * TPI=32 is required for N > 8192
   */
  /**
   * Larger takes longer to compile:
   * (32,32768) takes ~10 minutes (ecm was 6.0M)
   * (32,16384) takes ~2  minutes (ecm was 4.3M)
   * (16,8192) takes ~20 seconds (ecm was 4.3M)
   * (8, 1024) takes ~10 seconds (ecm was 3.7M)
   * GPU, No CGBN                (ecm was 3.5M)
   * No GPU, No CGBN             (ecm was 3.4M)
   */
  /**
   * TPI and BITS have to be set at compile time.
   * Adding multiple cgbn_params (and kernals) allows for dynamic selection
   * based on the size of N (e.g. N < 1024, N < 2048, N < 4096). If we go that
   * route it would be helpful to do that only during release builds.
   */
  const vector<uint32_t> available_kernels = {512, 1024, 2048, 8192};
  for (uint32_t kernel_bits : available_kernels) {
    tpi = kernel_bits <= 2048 ? 8 :
      kernel_bits <= 8192 ? 16 : 32;
  }


  //typedef cgbn_params_t<8, 512> cgbn_params_8_512;
  typedef cgbn_params_t<8, 1024> cgbn_params_8_1024;

  /**
   * TODO see if this can be replaced with uint32_t[5 * count * (BITS+31/32)*32]
   * then cast to mem_t in kernel to right type.
   */
  typedef typename curve_t<cgbn_params_8_1024>::mem_t mem_t;

  size_t    data_size = sizeof(mem_t) * 5 * curves;
  mem_t    *data, *gpu_data;

  int32_t   TPB=cgbn_params_8_1024::TPB;
  int32_t   TPI=cgbn_params_8_1024::TPI;
  int32_t   IPB=TPB/TPI; // IPB is instances per block

  size_t gpu_block_count = (curves+IPB-1)/IPB;

  // TODO change to take BITS, and have return uint32_t*
  data = curve_t<cgbn_params_8_1024>::set_p_2p(N, s, curves, ecm_params->sigma);

  // Copy data
  outputf (OUTPUT_VERBOSE, "Copying %d bits of data to GPU\n", data_size);
  CUDA_CHECK(hipMalloc((void **)&gpu_data, data_size));
  CUDA_CHECK(hipMemcpy(gpu_data, data, data_size, hipMemcpyHostToDevice));

  outputf (OUTPUT_VERBOSE, "Running GPU kernel<%ld,%d> TPI=%d...\n", gpu_block_count, TPB, TPI);
  kernel_double_add<cgbn_params_8_1024><<<gpu_block_count, TPB>>>(
      report, num_bits, gpu_s_bits, gpu_data, curves, ecm_params->sigma);

  /* error report uses managed memory, sync the device and check for cgbn errors */
  CUDA_CHECK(hipDeviceSynchronize());
  if (report->_error) {
    outputf (OUTPUT_ERROR, "\n\nerror: %d\n", report->_error);
  }
  CGBN_CHECK(report);

  /* gputime is measured in ms */
  CUDA_CHECK(hipEventRecord (stop, 0));
  CUDA_CHECK(hipEventSynchronize (stop));

  // Copy data back from GPU memory
  outputf (OUTPUT_VERBOSE, "Copying results back to CPU ...\n");
  CUDA_CHECK(hipMemcpy(data, gpu_data, data_size, hipMemcpyDeviceToHost));

  hipEventElapsedTime (gputime, start, stop);

  /**
   * This cast relies on mem_t (AKA struct cgbn_mem_t) being byte aligned without extra fields.
   * but it enables moving process_results out of kernel loop, without another copy of data
   */
  assert( sizeof(mem_t) == (cgbn_params_8_1024::BITS / 4) );
  youpi = process_results(
      factors, array_stage_found, N,
      (uint32_t *) data, cgbn_params_8_1024::BITS,
      curves, ecm_params->sigma);

  // clean up
  CUDA_CHECK(hipFree(gpu_s_bits));
  CUDA_CHECK(hipFree(gpu_data));
  CUDA_CHECK(cgbn_error_report_free(report));
  CUDA_CHECK(hipEventDestroy (start));
  CUDA_CHECK(hipEventDestroy (stop));

  free(s_bits);
  free(data);

  return youpi;
}
