#include "hip/hip_runtime.h"
/***

Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the "Software"),
to deal in the Software without restriction, including without limitation
the rights to use, copy, modify, merge, publish, distribute, sublicense,
and/or sell copies of the Software, and to permit persons to whom the
Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
IN THE SOFTWARE.

***/

#include "cgbn_stage1.h"

#include <cassert>
#include <stdio.h>
#include <stdlib.h>
#include <vector>

// GMP import must proceed cgbn.h
#include <gmp.h>
#include <cgbn.h>
#include <hip/hip_runtime.h>

#include "ecm.h"
#include "ecm-gpu.h"


void cuda_check(hipError_t status, const char *action=NULL, const char *file=NULL, int32_t line=0) {
  // check for cuda errors
  if (status!=hipSuccess) {
    fprintf (stderr, "CUDA error occurred: %s\n", hipGetErrorString(status));
    if (action!=NULL)
      fprintf (stderr, "While running %s   (file %s, line %d)\n", action, file, line);
    exit(1);
  }
}

// support routine copied from  "CGBN/samples/utility/support.h"
void cgbn_check(cgbn_error_report_t *report, const char *file=NULL, int32_t line=0) {
  // check for cgbn errors

  if(cgbn_error_report_check(report)) {
    fprintf (stderr, "\n");
    fprintf (stderr, "CGBN error occurred: %s\n", cgbn_error_string(report));

    if(report->_instance!=0xFFFFFFFF) {
      fprintf (stderr, "Error reported by instance %d", report->_instance);
      if(report->_blockIdx.x!=0xFFFFFFFF || report->_threadIdx.x!=0xFFFFFFFF)
        fprintf (stderr, ", ");
      if(report->_blockIdx.x!=0xFFFFFFFF)
      fprintf (stderr, "blockIdx=(%d, %d, %d) ", report->_blockIdx.x, report->_blockIdx.y, report->_blockIdx.z);
      if(report->_threadIdx.x!=0xFFFFFFFF)
        fprintf (stderr, "threadIdx=(%d, %d, %d)", report->_threadIdx.x, report->_threadIdx.y, report->_threadIdx.z);
      fprintf (stderr, "\n");
    }
    else {
      fprintf (stderr, "Error reported by blockIdx=(%d %d %d)", report->_blockIdx.x, report->_blockIdx.y, report->_blockIdx.z);
      fprintf (stderr, "threadIdx=(%d %d %d)\n", report->_threadIdx.x, report->_threadIdx.y, report->_threadIdx.z);
    }
    if(file!=NULL)
      fprintf (stderr, "file %s, line %d\n", file, line);
    exit(1);
  }
}

// Unify this with cudakernel.cu
#define CUDA_CHECK(action) cuda_check(action, #action, __FILE__, __LINE__)
#define CGBN_CHECK(report) cgbn_check(report, __FILE__, __LINE__)

static
void to_mpz(mpz_t r, const uint32_t *x, uint32_t count) {
  mpz_import(r, count, -1, sizeof(uint32_t), 0, 0, x);
}

static
void from_mpz(const mpz_t s, uint32_t *x, uint32_t count) {
  size_t words;

  if(mpz_sizeinbase(s, 2)>count*32) {
    fprintf (stderr, "from_mpz failed -- result does not fit\n");
    exit(1);
  }

  mpz_export(x, &words, -1, sizeof(uint32_t), 0, 0, s);
  while(words<count)
    x[words++]=0;
}



// ---------------------------------------------------------------- //

// See cgbn_error_t enum (cgbn.h:39)
#define cgbn_normalized_error ((cgbn_error_t) 14)
#define cgbn_positive_overflow ((cgbn_error_t) 15)
#define cgbn_negative_overflow ((cgbn_error_t) 16)

// Seems to adds very small overhead (1-10%)
#define VERIFY_NORMALIZED 1
// Adds even less overhead (<1%)
#define CHECK_ERROR 1

// Can dramatically change compile time
#if 1
    #define FORCE_INLINE __forceinline__
#else
    #define FORCE_INLINE
#endif


// The CGBN context uses the following three parameters:
//   TBP             - threads per block (zero means to use the blockDim.x)
//   MAX_ROTATION    - must be small power of 2, imperically, 4 works well
//   CONSTANT_TIME   - require constant time algorithms (currently, constant time algorithms are not available)

// Locally it will also be helpful to have several parameters:
//   TPI             - threads per instance
//   BITS            - number of bits per instance

const uint32_t TPB_DEFAULT = 128;

template<uint32_t tpi, uint32_t bits>
class cgbn_params_t {
  public:
  // parameters used by the CGBN context
  static const uint32_t TPB=TPB_DEFAULT;           // Reasonable default
  static const uint32_t MAX_ROTATION=4;            // good default value
  static const uint32_t SHM_LIMIT=0;               // no shared mem available
  static const bool     CONSTANT_TIME=false;       // not implemented

  // parameters used locally in the application
  static const uint32_t TPI=tpi;                   // threads per instance
  static const uint32_t BITS=bits;                 // instance size
};


template<class params>
class curve_t {
  public:

  typedef cgbn_context_t<params::TPI, params>   context_t;
  typedef cgbn_env_t<context_t, params::BITS>   env_t;
  typedef typename env_t::cgbn_t                bn_t;
  typedef cgbn_mem_t<params::BITS>              mem_t;

  context_t _context;
  env_t     _env;
  int32_t   _instance; // which curve instance is this

  // Constructor
  __device__ FORCE_INLINE curve_t(cgbn_monitor_t monitor, cgbn_error_report_t *report, int32_t instance) :
      _context(monitor, report, (uint32_t)instance), _env(_context), _instance(instance) {}

  // Verify 0 <= r < modulus
  __device__ FORCE_INLINE void assert_normalized(bn_t &r, const bn_t &modulus) {
    //if (VERIFY_NORMALIZED && _context.check_errors())
    if (VERIFY_NORMALIZED && CHECK_ERROR) {

        // Negative overflow
        if (cgbn_extract_bits_ui32(_env, r, params::BITS-1, 1)) {
            _context.report_error(cgbn_negative_overflow);
        }
        // Positive overflow
        if (cgbn_compare(_env, r, modulus) >= 0) {
            _context.report_error(cgbn_positive_overflow);
        }
    }
  }

  // Normalize after addition
  __device__ FORCE_INLINE void normalize_addition(bn_t &r, const bn_t &modulus) {
      if (cgbn_compare(_env, r, modulus) >= 0) {
          cgbn_sub(_env, r, r, modulus);
      }
  }

  // Normalize after subtraction (handled instead by checking carry)
  /*
  __device__ FORCE_INLINE void normalize_subtraction(bn_t &r, const bn_t &modulus) {
      if (cgbn_extract_bits_ui32(_env, r, params::BITS-1, 1)) {
          cgbn_add(_env, r, r, modulus);
      }
  }
  */

  /**
   * Calculate (r * m) / 2^32 mod modulus
   *
   * This removes a factor of 2^32 which is not present in m.
   * Otherwise m (really d) needs to be passed as a bigint not a uint32
   */
  __device__ FORCE_INLINE void special_mult_ui32(bn_t &r, uint32_t m, const bn_t &modulus, uint32_t np0) {
    //uint32_t thread_i = (blockIdx.x*blockDim.x + threadIdx.x)%params::TPI;
    bn_t temp;

    uint32_t carry_t1 = cgbn_mul_ui32(_env, r, r, m);
    uint32_t t1_0 = cgbn_extract_bits_ui32(_env, r, 0, 32);
    uint32_t q = t1_0 * np0;
    uint32_t carry_t2 = cgbn_mul_ui32(_env, temp, modulus, q);

    cgbn_shift_right(_env, r, r, 32);
    cgbn_shift_right(_env, temp, temp, 32);
    // Add back overflow carry
    cgbn_insert_bits_ui32(_env, r, r, params::BITS-32, 32, carry_t1);
    cgbn_insert_bits_ui32(_env, temp, temp, params::BITS-32, 32, carry_t2);

    // This needs to be measured at block containing top bit of modulus
    int32_t carry_q = cgbn_add(_env, r, r, temp);
    carry_q += cgbn_add_ui32(_env, r, r, t1_0 != 0); // add 1
    while (carry_q != 0) {
        carry_q -= cgbn_sub(_env, r, r, modulus);
    }

    // 0 <= r, temp < modulus => r + temp + 1 < 2*modulus
    if (cgbn_compare(_env, r, modulus) >= 0) {
        cgbn_sub(_env, r, r, modulus);
    }
  }


  __device__ FORCE_INLINE void double_add_v2(
          bn_t &q, bn_t &u,
          bn_t &w, bn_t &v,
          uint32_t d,
          uint32_t bit_number,
          const bn_t &modulus,
          const uint32_t np0) {
    // q = xA = aX
    // u = zA = aY
    // w = xB = bX
    // v = zB = bY

    /* Doesn't seem to be a large cost to using many extra variables */
    bn_t t, CB, DA, AA, BB, K, dK;

    /* Can maybe use one more bit if cgbn_add subtracts when carry happens */

    cgbn_add(_env, t, v, w); // t = (bY + bX)
    normalize_addition(t, modulus);
    if (cgbn_sub(_env, v, v, w)) // v = (bY - bX)
        cgbn_add(_env, v, v, modulus);


    cgbn_add(_env, w, u, q); // w = (aY + aX)
    normalize_addition(w, modulus);
    if (cgbn_sub(_env, u, u, q)) // u = (aY - aX)
        cgbn_add(_env, u, u, modulus);
    if (VERIFY_NORMALIZED) {
        assert_normalized(t, modulus);
        assert_normalized(v, modulus);
        assert_normalized(w, modulus);
        assert_normalized(u, modulus);
    }

    cgbn_mont_mul(_env, CB, t, u, modulus, np0); // C*B
        normalize_addition(CB, modulus);
    cgbn_mont_mul(_env, DA, v, w, modulus, np0); // D*A
        normalize_addition(DA, modulus);

    cgbn_mont_sqr(_env, AA, w, modulus, np0);    // AA
    cgbn_mont_sqr(_env, BB, u, modulus, np0);    // BB
    normalize_addition(AA, modulus);
    normalize_addition(BB, modulus);
    if (VERIFY_NORMALIZED) {
        assert_normalized(CB, modulus);
        assert_normalized(DA, modulus);
        assert_normalized(AA, modulus);
        assert_normalized(BB, modulus);
    }

    // q = aX is finalized
    cgbn_mont_mul(_env, q, AA, BB, modulus, np0); // AA*BB
        normalize_addition(q, modulus);
        assert_normalized(q, modulus);

    if (cgbn_sub(_env, K, AA, BB)) // K = AA-BB
        cgbn_add(_env, K, K, modulus);

    // By definition of d = (sigma / 2^32) % MODN
    // K = k*R
    // dK = d*k*R = (K * R * sigma) >> 32
    cgbn_set(_env, dK, K);
    special_mult_ui32(dK, d, modulus, np0); // dK = K*d
        assert_normalized(dK, modulus);

    cgbn_add(_env, u, BB, dK); // BB + dK
    normalize_addition(u, modulus);
    if (VERIFY_NORMALIZED) {
        assert_normalized(K, modulus);
        assert_normalized(dK, modulus);
        assert_normalized(u, modulus);
    }

    // u = aY is finalized
    cgbn_mont_mul(_env, u, K, u, modulus, np0); // K(BB+dK)
        normalize_addition(u, modulus);
        assert_normalized(u, modulus);

    cgbn_add(_env, w, DA, CB); // DA + CB
    normalize_addition(w, modulus);
    if (cgbn_sub(_env, v, DA, CB)) // DA - CB
        cgbn_add(_env, v, v, modulus);
    if (VERIFY_NORMALIZED) {
        assert_normalized(w, modulus);
        assert_normalized(v, modulus);
    }

    // w = bX is finalized
    cgbn_mont_sqr(_env, w, w, modulus, np0); // (DA+CB)^2 mod N
        normalize_addition(w, modulus);
        assert_normalized(w, modulus);

    cgbn_mont_sqr(_env, v, v, modulus, np0); // (DA-CB)^2 mod N
        normalize_addition(v, modulus);
        assert_normalized(v, modulus);

    // v = bY is finalized
    cgbn_shift_left(_env, v, v, 1); // double
    normalize_addition(v, modulus);
        assert_normalized(v, modulus);
  }
};


static
uint32_t* set_p_2p(const mpz_t N, const mpz_t s,
                   uint32_t curves, uint32_t sigma,
                   uint32_t BITS, size_t *data_size) {
  // P1_x, P1_y = (2,1)
  // 2P_x, 2P_y = (9, 64 * d + 8)

  /** Consider curve copies of N (AKA modulo) */
  const size_t limbs_per = BITS/32;
  *data_size = 5 * curves * limbs_per * sizeof(uint32_t);
  uint32_t *data = (uint32_t*) malloc(*data_size);
  uint32_t *datum = data;

  mpz_t x;
  mpz_init(x);
  for(int index = 0; index < curves; index++) {
      // d = (sigma / 2^32) mod N BUT 2^32 handled by special_mul_ui32
      uint32_t d = sigma + index;

      // mod
      from_mpz(N, datum + 0 * limbs_per, BITS/32);

      // P1 (X, Y)
      mpz_set_ui(x, 2);
      from_mpz(x, datum + 1 * limbs_per, BITS/32);
      mpz_set_ui(x, 1);
      from_mpz(x, datum + 2 * limbs_per, BITS/32);

      // 2P = P2 (X, Y)
      // P2_y = 64 * d + 8
      mpz_set_ui(x, 9);
      from_mpz(x, datum + 3 * limbs_per, BITS/32);

      // d = sigma * mod_inverse(2 ** 32, N)
      mpz_ui_pow_ui(x, 2, 32);
      mpz_invert(x, x, N);
      mpz_mul_ui(x, x, d);
      // P2_y = 64 * d - 2;
      mpz_mul_ui(x, x, 64);
      mpz_add_ui(x, x, 8);
      mpz_mod(x, x, N);

      outputf (OUTPUT_TRACE, "sigma %d => P2_y: %Zd\n", d, x); 
      from_mpz(x, datum + 4 * limbs_per, BITS/32);
      datum += 5 * limbs_per;
  }
  mpz_clear(x);
  return data;
}


// kernel implementation using cgbn
template<class params>
__global__ void kernel_double_add(
        cgbn_error_report_t *report,
        uint32_t s_bits,
        char* gpu_s_bits,
        uint32_t *data,
        uint32_t count,
        uint32_t sigma_0) {
  // decode an instance_i number from the blockIdx and threadIdx
  int32_t instance_i = (blockIdx.x*blockDim.x + threadIdx.x)/params::TPI;
  if(instance_i >= count)
    return;

  /* Cast uint32_t array to mem_t */
  typename curve_t<params>::mem_t *data_cast = (typename curve_t<params>::mem_t*) data;

  cgbn_monitor_t monitor = CHECK_ERROR ? cgbn_report_monitor : cgbn_no_checks;

  curve_t<params> curve(monitor, report, instance_i);
  typename curve_t<params>::bn_t  aX, aY, bX, bY, modulus;
 
  uint32_t np0;
  { // Setup
      cgbn_load(curve._env, modulus, &data_cast[5*instance_i+0]);
      cgbn_load(curve._env, aX, &data_cast[5*instance_i+1]);
      cgbn_load(curve._env, aY, &data_cast[5*instance_i+2]);
      cgbn_load(curve._env, bX, &data_cast[5*instance_i+3]);
      cgbn_load(curve._env, bY, &data_cast[5*instance_i+4]);

      // Convert everything to mont
      np0 = cgbn_bn2mont(curve._env, aX, aX, modulus);
      cgbn_bn2mont(curve._env, aY, aY, modulus);
      cgbn_bn2mont(curve._env, bX, bX, modulus);
      cgbn_bn2mont(curve._env, bY, bY, modulus);

      {
        curve.assert_normalized(aX, modulus);
        curve.assert_normalized(aY, modulus);
        curve.assert_normalized(bX, modulus);
        curve.assert_normalized(bY, modulus);
      }
  }

  uint32_t d = sigma_0 + instance_i;

  for (int b = s_bits; b > 0; b--) {
    /**
     * TODO generates a lot of duplicate inlined code, not sure how to improve
     * Tried with swappings pointers (with a single call to double_add_v2)
     */
    if (gpu_s_bits[s_bits - b] == 0) {
        curve.double_add_v2(aX, aY, bX, bY, d, b, modulus, np0);
    } else {
        curve.double_add_v2(bX, bY, aX, aY, d, b, modulus, np0);
    }
  }

  { // Final output
    // Convert everything back to bn
    cgbn_mont2bn(curve._env, aX, aX, modulus, np0);
    cgbn_mont2bn(curve._env, aY, aY, modulus, np0);
    cgbn_mont2bn(curve._env, bX, bX, modulus, np0);
    cgbn_mont2bn(curve._env, bY, bY, modulus, np0);
    {
      curve.assert_normalized(aX, modulus);
      curve.assert_normalized(aY, modulus);
      curve.assert_normalized(bX, modulus);
      curve.assert_normalized(bY, modulus);
    }
    cgbn_store(curve._env, &data_cast[5*instance_i+1], aX);
    cgbn_store(curve._env, &data_cast[5*instance_i+2], aY);
    cgbn_store(curve._env, &data_cast[5*instance_i+3], bX);
    cgbn_store(curve._env, &data_cast[5*instance_i+4], bY);
  }
}

static
int findfactor(mpz_t factor, const mpz_t N, const mpz_t x_final, const mpz_t y_final) {
    // XXX: combine / refactor logic with cudawrapper.c findfactor

    mpz_t temp;
    mpz_init(temp);

    // Check if factor found

    bool inverted = mpz_invert(temp, y_final, N);    // aY ^ (N-2) % N
    if (inverted) {
        mpz_mul(temp, x_final, temp);         // aX * aY^-1
        mpz_mod(factor, temp, N);             // "Residual"
        mpz_clear(temp);
        return ECM_NO_FACTOR_FOUND;
    }
    mpz_clear(temp);

    mpz_gcd(factor, y_final, N);
    return ECM_FACTOR_FOUND_STEP1;
}


static
int verify_size_of_n(const mpz_t N, size_t max_bits) {
  size_t n_log2 = mpz_sizeinbase(N, 2);

  // using check_gpuecm.sage it looks like 4 bits would suffice
  size_t max_usable_bits = max_bits - 6;

  if (n_log2 <= max_usable_bits)
    return ECM_NO_FACTOR_FOUND;

  outputf (OUTPUT_ERROR, "GPU: N(%d bits) + carry(%d bits) > BITS(%d)\n",
      n_log2, max_bits - max_usable_bits, max_bits);
  outputf (OUTPUT_ERROR, "GPU: Error, input number should be stricly lower than 2^%d\n",
      max_usable_bits);
  return ECM_ERROR;
}


static
char* allocate_and_set_s_bits(const mpz_t s, int *sbits) {
  uint32_t num_bits = *sbits = mpz_sizeinbase(s, 2) - 1;
  assert( 1 <= num_bits <= 100'000'000 );

  // Use int* so that size can be stored in first element, could pass around extra size.
  char *s_bits = (char*) malloc(sizeof(char) * num_bits);
  for (int i = 0; i < num_bits; i++) {
      s_bits[i] = mpz_tstbit (s, num_bits - 1 - i);
  }
  return s_bits;
}

static
int process_results(mpz_t *factors, int *array_stage_found,
                    const mpz_t N,
                    const uint32_t *data, uint32_t cgbn_bits,
                    int curves, uint32_t sigma) {
  mpz_t x_final, y_final, modulo;
  mpz_init(modulo);
  mpz_init(x_final);
  mpz_init(y_final);

  const uint32_t limbs_per = cgbn_bits / 32;

  int youpi = ECM_NO_FACTOR_FOUND;
  for(size_t i = 0; i < curves; i++) {
    const uint32_t *datum = data + (5 * i * limbs_per);;

    if (test_verbose (OUTPUT_TRACE) && i == 0) {
      to_mpz(modulo, datum + 0 * limbs_per, limbs_per);
      outputf (OUTPUT_TRACE, "index: 0 modulo: %Zd\n", modulo);

      to_mpz(x_final, datum + 1 * limbs_per, limbs_per);
      to_mpz(y_final, datum + 2 * limbs_per, limbs_per);
      outputf (OUTPUT_TRACE, "index: 0 pA: (%Zd, %Zd)\n", x_final, y_final);

      to_mpz(x_final, datum + 3 * limbs_per, limbs_per);
      to_mpz(y_final, datum + 4 * limbs_per, limbs_per);
      outputf (OUTPUT_TRACE, "index: 0 pB: (%Zd, %Zd)\n", x_final, y_final);
    }

    // Make sure we were testing the right number.
    to_mpz(modulo, datum + 0 * limbs_per, limbs_per);
    assert(mpz_cmp(modulo, N) == 0);

    to_mpz(x_final, datum + 1 * limbs_per, limbs_per);
    to_mpz(y_final, datum + 2 * limbs_per, limbs_per);

    array_stage_found[i] = findfactor(factors[i], N, x_final, y_final);
    if (array_stage_found[i] != ECM_NO_FACTOR_FOUND) {
      youpi = array_stage_found[i];
      outputf (OUTPUT_NORMAL, "GPU: factor %Zd found in Step 1 with curve %ld (-sigma %d:%d)\n",
          factors[i], i, ECM_PARAM_BATCH_32BITS_D, sigma + i);
    }
  }
  
  mpz_init(modulo);
  mpz_clear(x_final);
  mpz_clear(y_final);

  return youpi;
}

int run_cgbn(mpz_t *factors, int *array_stage_found,
             const mpz_t N, const mpz_t s, float *gputime,
             ecm_params_t *ecm_params) {

  const size_t MAX_BITS = 1024;

  size_t curves = ecm_params->curves;  
  assert( ecm_params->sigma > 0 );
  assert( ((uint64_t) ecm_params->sigma + curves) <= 0xFFFFFFFF ); // no overflow

  /* Validate N's size */
  int youpi = verify_size_of_n(N, MAX_BITS);
  if (youpi != ECM_NO_FACTOR_FOUND) {
    return youpi;
  }

  int s_num_bits;
  char *s_bits = allocate_and_set_s_bits(s, &s_num_bits);
  assert( 1 <= s_num_bits <= 100'000'000 );
  assert( s_bits != NULL );

  // Keeps CPU from busy waiting during GPU execution.
  CUDA_CHECK(hipSetDeviceFlags (hipDeviceScheduleBlockingSync));
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate (&start));
  CUDA_CHECK(hipEventCreate (&stop));
  CUDA_CHECK(hipEventRecord (start, 0));

  // Copy s_bits
  char     *gpu_s_bits;
  CUDA_CHECK(hipMalloc((void **)&gpu_s_bits, sizeof(char) * s_num_bits));
  CUDA_CHECK(hipMemcpy(gpu_s_bits, s_bits, sizeof(char) * s_num_bits, hipMemcpyHostToDevice));


  cgbn_error_report_t *report;
  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK(cgbn_error_report_alloc(&report));


  size_t    data_size;
  uint32_t *data, *gpu_data;

  uint32_t  BITS = 0;        // kernel bits
  int32_t   TPB=TPB_DEFAULT; // Always the same default
  int32_t   TPI;
  int32_t   IPB;             // IPB = TPB / TPI, instances per block
  size_t    BLOCK_COUNT;     // How many blocks to cover all curves

  /**
   * Smaller TPI (e.g. 8) is faster (TPI=4 seems worse than TPI=8).
   * Larger TPI (e.g. 32) for running a single curve (or large N).
   * TPI=8  is required for N > 512
   * TPI=16 is required for N > 2048
   * TPI=32 is required for N > 8192
   */
  /**
   * Larger takes longer to compile:
   * (32,32768) takes ~10 minutes (ecm was 6.0M)
   * (32,16384) takes ~2  minutes (ecm was 4.3M)
   * (16,8192) takes ~20 seconds (ecm was 4.3M)
   * (8, 1024) takes ~10 seconds (ecm was 3.7M)
   * GPU, No CGBN                (ecm was 3.5M)
   * No GPU, No CGBN             (ecm was 3.4M)
   */
  /**
   * TPI and BITS have to be set at compile time.
   * Adding multiple cgbn_params (and kernals) allows for dynamic selection
   * based on the size of N (e.g. N < 1024, N < 2048, N < 4096). If we go that
   * route it would be helpful to do that only during release builds.
   */

  typedef cgbn_params_t<8, 1024>  cgbn_params_8_1024;
  typedef cgbn_params_t<8, 512>   cgbn_params_8_512;
#ifdef IS_DEV_BUILD
  const std::vector<uint32_t> available_kernels = { 1024 };
#else
  typedef cgbn_params_t<8, 1024>  cgbn_params_8_1024;
  typedef cgbn_params_t<8, 1536>  cgbn_params_8_1536;
  typedef cgbn_params_t<8, 2048>  cgbn_params_8_2048;
  typedef cgbn_params_t<16, 3072> cgbn_params_16_3072;
  const std::vector<uint32_t> available_kernels = { 512, 1024, 1536, 2048, 3072 };
#endif /* IS_DEV_BUILD */

  for (uint32_t kernel_bits : available_kernels) {
    if (kernel_bits + 6 >=  mpz_sizeinbase(N, 2)) {
      BITS = kernel_bits;
      assert( BITS % 32 == 0 );
      TPI = (BITS <= 2048) ? 8 : ((BITS <= 8192) ? 16 : 32);
      IPB = TPB / TPI;
      BLOCK_COUNT = (curves + IPB - 1) / IPB;
      break;
    }
  }
  if (BITS == 0) {
    outputf (OUTPUT_ERROR, "No available CGBN Kernel large enough to process N(%d bits)\n",
        mpz_sizeinbase(N, 2));
    return ECM_ERROR;
  }

  /** Relies on mem_t (AKA struct cgbn_mem_t) being byte aligned without extra fields. */
  assert( sizeof(curve_t<cgbn_params_8_512>::mem_t) == 512/8 );
  assert( sizeof(curve_t<cgbn_params_8_1024>::mem_t) == 1024/8 );
  data = set_p_2p(N, s, curves, ecm_params->sigma, BITS, &data_size);

  // Copy data
  outputf (OUTPUT_VERBOSE, "Copying %d bits of data to GPU\n", data_size);
  CUDA_CHECK(hipMalloc((void **)&gpu_data, data_size));
  CUDA_CHECK(hipMemcpy(gpu_data, data, data_size, hipMemcpyHostToDevice));

  outputf (OUTPUT_VERBOSE, "Running CGBN<%d,%d> kernel<%ld,%d>...\n", BITS, TPI, BLOCK_COUNT, TPB);
  if (BITS == 512) {
    kernel_double_add<cgbn_params_8_512><<<BLOCK_COUNT, TPB>>>(report, s_num_bits, gpu_s_bits, gpu_data, curves, ecm_params->sigma);
  } else if (BITS == 1024) {
    kernel_double_add<cgbn_params_8_1024><<<BLOCK_COUNT, TPB>>>(report, s_num_bits, gpu_s_bits, gpu_data, curves, ecm_params->sigma);
#ifndef IS_DEV_BUILD
  } else if (BITS == 1536) {
    kernel_double_add<cgbn_params_8_1536><<<BLOCK_COUNT, TPB>>>(report, s_num_bits, gpu_s_bits, gpu_data, curves, ecm_params->sigma);
  } else if (BITS == 2048) {
    kernel_double_add<cgbn_params_8_2048><<<BLOCK_COUNT, TPB>>>(report, s_num_bits, gpu_s_bits, gpu_data, curves, ecm_params->sigma);
  } else if (BITS == 3072) {
    kernel_double_add<cgbn_params_16_3072><<<BLOCK_COUNT, TPB>>>(report, s_num_bits, gpu_s_bits, gpu_data, curves, ecm_params->sigma);
#endif
  } else {
    outputf (OUTPUT_ERROR, "CGBN Kernel not found for %d bits\n", BITS);
    return ECM_ERROR;
  }

  /* error report uses managed memory, sync the device and check for cgbn errors */
  CUDA_CHECK(hipDeviceSynchronize());
  if (report->_error)
    outputf (OUTPUT_ERROR, "\n\nerror: %d\n", report->_error);
  CGBN_CHECK(report);

  /* gputime is measured in ms */
  CUDA_CHECK(hipEventRecord (stop, 0));
  CUDA_CHECK(hipEventSynchronize (stop));

  // Copy data back from GPU memory
  outputf (OUTPUT_VERBOSE, "Copying results back to CPU ...\n");
  CUDA_CHECK(hipMemcpy(data, gpu_data, data_size, hipMemcpyDeviceToHost));

  hipEventElapsedTime (gputime, start, stop);

  youpi = process_results(
      factors, array_stage_found, N,
      data, BITS,
      curves, ecm_params->sigma);

  // clean up
  CUDA_CHECK(hipFree(gpu_s_bits));
  CUDA_CHECK(hipFree(gpu_data));
  CUDA_CHECK(cgbn_error_report_free(report));
  CUDA_CHECK(hipEventDestroy (start));
  CUDA_CHECK(hipEventDestroy (stop));

  free(s_bits);
  free(data);

  return youpi;
}
