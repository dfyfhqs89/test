#include "hip/hip_runtime.h"
#include "cudaarith.h"

//specific functions for compute capability 1.3
//#ifdef CC13
//  (A > B)?, returns 1(true), -1(false) or 0(a=b) 
//Assume A and B are normalize (no carry or borrow)
__device__ int Cuda_Cmp(const biguint_t A, const biguint_t B)
{
	unsigned int i = SIZE_NUMBER-1;
	do
	{
		if (A[i] > B[i])
			return 1;
		if (A[i] < B[i])
			return -1;
		i--;
	}while(i!=-1);
	return 0;
}
//#endif
/*
//specific functions for compute capability 20
#ifdef CC20
//  (A > B)?, returns 1(true), -1(false) or 0(a=b) 
//Assume A and B are normalize (no carry or borrow)
__device__ int Cuda_Is_Eq(const biguint_t A, const biguint_t B)
{
	return A[threadIdx.x]==B[threadIdx.x];
}
__device__ int Cuda_Is_Gt(const biguint_t A, const biguint_t B)
{
	return A[threadIdx.x]>B[threadIdx.x];
}
__device__ int Cuda_Cmp(const biguint_t A, const biguint_t B)
{
	unsigned int ballot=__ballot(Cuda_Is_Eq(A,B));
	if (ballot==0)
		return 0;
	else
		if (__ballot(Cuda_Is_Gt(A,B))>__ballot(Cuda_Is_Gt(B,A)))
			return 1;
		else
			return -1;
}
#endif
*/

//return -1 if A<cst, 0 if A=cst, 1 if A>cst
__device__ int Cuda_ge_cst(const biguint_t A, const biguint_t cst, const int cy)
{
	unsigned int i = SIZE_NUMBER-1;
	if (cy>0)
		return 1;
	do
	{
		if (A[i] > cst[i])
			return 1;
		if (A[i] < cst[i])
			return -1;
		i--;
	}while(i!=0);
	return 0;
}

//Normalise a result; 
__device__ int Cuda_Is_Normalize(dbigint_t cy)
{
	if (threadIdx.x==SIZE_NUMBER-1 || cy[threadIdx.x]==0)
		return 0;
	else
		return 1;
}

__device__ void Cuda_Normalize(biguint_t A,dbigint_t cy)
{
	//if (threadIdx.x==0)
	//	cptnormalize[blockIdx.x]++;

	int oldcy;
	if (threadIdx.x==0)
		oldcy = 0;
	else
	{
		oldcy = cy[threadIdx.x-1];
		cy[threadIdx.x-1]=0;
	}

	if (oldcy>=0)
	{
		//Cuda_Add1(A,cy,oldcy);
		A[threadIdx.x]+=oldcy;
		cy[threadIdx.x]+=(A[threadIdx.x]<oldcy);
	}
	else // oldcy < 0
	{
		//Cuda_Sub1(A,cy,-oldcy);
		cy[threadIdx.x]-=(A[threadIdx.x]< -oldcy);
		A[threadIdx.x]-= -oldcy;
	}
}

__device__ void Cuda_Fully_Normalize(biguint_t A,dbigint_t cy)
{
	do
	{
	Cuda_Normalize(A,cy);
	}while(__any(Cuda_Is_Normalize(cy))!=0);
}


__device__ void Cuda_Add (biguint_t r, dbigint_t cy ,const biguint_t b)
{
	r[threadIdx.x]+=b[threadIdx.x];
	cy[threadIdx.x]+=(r[threadIdx.x]<b[threadIdx.x]);
}

//Assume r and b are different; r and a can be the same.
__device__ void Cuda_Add 
(biguint_t r, dbigint_t cy ,const biguint_t a, const biguint_t b)
{
	r[threadIdx.x]=a[threadIdx.x];
	r[threadIdx.x]+=b[threadIdx.x];
	cy[threadIdx.x]+=(r[threadIdx.x]<b[threadIdx.x]);
}

__device__ void Cuda_Sub (biguint_t r, dbigint_t cy, const biguint_t b)
{
	cy[threadIdx.x]-=(r[threadIdx.x] < b[threadIdx.x]);
	r[threadIdx.x] -= b[threadIdx.x];
}

//Assume r and b are different; r and a can be the same.
__device__ void Cuda_Sub 
(biguint_t r, dbigint_t cy, const biguint_t a, const biguint_t b)
{
	r[threadIdx.x]=a[threadIdx.x];
	cy[threadIdx.x]-=(r[threadIdx.x] < b[threadIdx.x]);
	r[threadIdx.x] -= b[threadIdx.x];
}

//Compute Rmod <- A + B [mod] 
__device__ void Cuda_Add_mod
(biguint_t Rmod, dbigint_t cy, const biguint_t A,const biguint_t B,const biguint_t mod)
{
  Cuda_Add(Rmod,cy,A,B);
	Cuda_Fully_Normalize(Rmod,cy);	
  
  if (Cuda_ge_cst (Rmod,mod,cy[SIZE_NUMBER-1]) >= 0) // (a >= mod)? 
	{
   	Cuda_Sub (Rmod, cy, mod); // R <- R - mod 
		Cuda_Fully_Normalize(Rmod,cy);	
	}
}

//Compute Rmod  <-Rmod + A [mod] 
__device__ void Cuda_Add_mod
(biguint_t Rmod, dbigint_t cy, const biguint_t A,const biguint_t mod)
{
 	Cuda_Add(Rmod,cy,A);
	Cuda_Fully_Normalize(Rmod,cy);	

  if (Cuda_ge_cst (Rmod,mod,cy[SIZE_NUMBER-1]) >= 0) // (a >= mod)? 
	{
   	Cuda_Sub (Rmod, cy, mod);  
		Cuda_Fully_Normalize(Rmod,cy);	
	}
}


//Compute Rmod <- A - B [mod] 
__device__ void Cuda_Sub_mod 
(biguint_t Rmod, dbigint_t cy, const biguint_t A,const biguint_t B,const biguint_t mod)
{
 	Cuda_Sub(Rmod,cy,A,B);
	Cuda_Fully_Normalize(Rmod,cy);	
	
 	if (cy[SIZE_NUMBER-1] <0 ) // we should subtract 1 at smod[n] 
 	{
   	Cuda_Add (Rmod, cy, mod);
		Cuda_Fully_Normalize(Rmod,cy);	
 	}
}

//Compute Rmod <- Rmod - A [mod]
__device__ void Cuda_Sub_mod 
(biguint_t Rmod, dbigint_t cy, const biguint_t A, const biguint_t mod)
{
 	Cuda_Sub(Rmod,cy,A);
	Cuda_Fully_Normalize(Rmod,cy);	

 	if (cy[SIZE_NUMBER-1] <0 ) 
 	{
   	Cuda_Add (Rmod, cy, mod);
		Cuda_Fully_Normalize(Rmod,cy);	
 	}
}

//  Return h, l such that h*2^32 + l = A*B 
__device__ void Cuda_Mul_uint (unsigned int *h, unsigned int *l, const unsigned int A,const unsigned int B)
{
		*h=__umulhi(A,B);
		*l=A*B;
}

__device__ void Cuda_Mul
(dbiguint_t R, dbigint_t cy, const biguint_t A,const biguint_t B)
{
	int i;
	unsigned int h,l;
	
	for (i=0;i<SIZE_NUMBER;i++)
	{
		//h*2^32+l =A[i]*B[threadIDx.x]
		Cuda_Mul_uint(&h,&l,A[threadIdx.x],B[i]);
		
		R[i+threadIdx.x] +=l;
		cy[i+threadIdx.x]+=(R[i+threadIdx.x] < l);
		
		R[i+1+threadIdx.x] +=h;
		cy[i+1+threadIdx.x]+=(R[i+1+threadIdx.x]<h);
	}

	//Normalize : but R and cy are 2 * SIZE_NUMBER long
	Cuda_Fully_Normalize(R,cy);
	Cuda_Fully_Normalize(R+SIZE_NUMBER-1,cy+SIZE_NUMBER-1);

	//if (threadIdx.x==0)
	//	cptnormalizemul[blockIdx.x]++;

	if (threadIdx.x==0)
	{
		R[2*SIZE_NUMBER-1]+=cy[2*SIZE_NUMBER-2];
		cy[2*SIZE_NUMBER-2]=0;//only to let cy clean in order to re-use it
	}
}

__device__ void Cuda_Square (dbiguint_t R, dbigint_t cy, const biguint_t A)
{
	Cuda_Mul(R,cy,A,A);
}

//Assume r<N^2
__device__ void Cuda_RedMontgomery (biguint_t mul, dbigint_t cy, const biguint_t mod, dbiguint_t r, dbiguint_t temp)
{
	//temp=((r mod 2^(32*SIZE_NUMBER))*mod^-1) (mod^-1 already compute)
	Cuda_Mul(temp,cy,r,invmodcst);//pour r que la partie mod R compte : Ok
	
	//mul = temp (mod 2^(32*SIZE_NUMBER)) 
	mul[threadIdx.x]=temp[threadIdx.x];
	temp[threadIdx.x]=0;
	temp[threadIdx.x+SIZE_NUMBER]=0;
	
	//temp=mul*m
	Cuda_Mul(temp,cy,mul,mod);
	
	//r=r+temp // r and temp2 are 2*SIZE_NUMBER long
	Cuda_Add(r,cy,temp);
	Cuda_Add(r+SIZE_NUMBER,cy+SIZE_NUMBER,temp+SIZE_NUMBER);
	//Normalize : but R and cy are 2 * SIZE_NUMBER long
	Cuda_Fully_Normalize(r,cy);
	Cuda_Fully_Normalize(r+SIZE_NUMBER-1,cy+SIZE_NUMBER-1);

	if (threadIdx.x==0)
		{
		r[2*SIZE_NUMBER-1]+=cy[2*SIZE_NUMBER-2];
		cy[2*SIZE_NUMBER-2]=0;//only to let cy clean in order to re-use it
		}

	//return r/ 2^(32*SIZE_NUMBER)
	mul[threadIdx.x]=r[threadIdx.x+SIZE_NUMBER];

	if (Cuda_Cmp (mul,mod) >= 0) // (mul >= mod)? 
	{
  	Cuda_Sub (mul, cy, mod); 
		Cuda_Fully_Normalize(mul,cy);	
	}
}

//Assume A ans B are the montgomery representation
//Compute mul = A * B * 2^-(32*SIZE_NUMBER) mod[mod]
// r and temp have size 2*SIZE_NUMBER
__device__ void Cuda_Mul_mod (biguint_t mul, dbigint_t cy, const biguint_t A,const biguint_t B, const biguint_t mod, dbiguint_t r, dbiguint_t temp)
{
	temp[threadIdx.x]=0;
	temp[threadIdx.x+SIZE_NUMBER]=0;
	r[threadIdx.x]=0;
	r[threadIdx.x+SIZE_NUMBER]=0;
	
	//__syncthreads();
	//r=a*b
	Cuda_Mul(r,cy,A,B);
	Cuda_RedMontgomery (mul, cy, mod, r, temp);
}


//Assume A ans B are the montgomery representation
//Compute mul = A * A * 2^-(32*SIZE_NUMBER) mod[mod]
// r and temp have size 2*SIZE_NUMBER
__device__ void Cuda_Square_mod (biguint_t mul, dbigint_t cy, const biguint_t A, const biguint_t mod, dbiguint_t r, dbiguint_t temp)
{
	temp[threadIdx.x]=0;
	temp[threadIdx.x+SIZE_NUMBER]=0;
	r[threadIdx.x]=0;
	r[threadIdx.x+SIZE_NUMBER]=0;
	
	//__syncthreads();
	//r=a*b
	Cuda_Square(r,cy,A);
	Cuda_RedMontgomery (mul, cy, mod, r, temp);
}

__device__ void Cuda_Ell_Dbl(biguint_t x2p, biguint_t z2p, const biguint_t xp, const biguint_t zp, const biguint_t N, const biguint_t d, biguint_t temp_u, biguint_t temp_v, dbiguint_t temp_r, dbiguint_t temp_r2, dbigint_t cy)
{
	//u<-xp+zp mod N
	Cuda_Add_mod(temp_u,cy,xp,zp,N);
	//u <- u^2
	Cuda_Square_mod(temp_u,cy,temp_u,N,temp_r,temp_r2);
	//v<-xp-zp mod N
	Cuda_Sub_mod(temp_v,cy,xp,zp,N);
	//v <- v^2
	Cuda_Square_mod(temp_v,cy,temp_v,N,temp_r,temp_r2);
	//x2p=u-v mod N    x2p is used as a temporary variable here
	Cuda_Sub_mod(x2p,cy,temp_u,temp_v,N);
	//z2p<-x2p*d mod N z2p is used as a temporary variable here
	Cuda_Mul_mod(z2p,cy,x2p,d,N,temp_r,temp_r2);
	//z2p<- z2p+v mod N
	Cuda_Add_mod(z2p,cy,temp_v,N);
	//z2p <- x2p*z2p
	Cuda_Mul_mod(z2p,cy,x2p,z2p,N,temp_r,temp_r2);
	//x2p <- u*v
	Cuda_Mul_mod(x2p,cy,temp_u,temp_v,N,temp_r,temp_r2);
}

__device__ void Cuda_Ell_Add(biguint_t xplus, biguint_t zplus, const biguint_t xp, const biguint_t zp, const biguint_t xq, const biguint_t zq, const biguint_t xminus, const biguint_t zminus, const biguint_t N, biguint_t temp_u, biguint_t temp_v, biguint_t temp_w, dbiguint_t temp_r, dbiguint_t temp_r2, dbigint_t cy)
{
	unsigned int tmp;
	
	//u<-xp+zp mod N
	Cuda_Add_mod(temp_u,cy,xp,zp,N);
	//v<-xq-zq mod N
	Cuda_Sub_mod(temp_v,cy,xq,zq,N);
	//v<-u*v mod N
	Cuda_Mul_mod(temp_v,cy,temp_u,temp_v,N,temp_r,temp_r2);
	//u<-xp-zp mod N 
	Cuda_Sub_mod(temp_u,cy,xp,zp,N);
	//w<-xq+zq mod N
	Cuda_Add_mod(temp_w,cy,zq,xq,N);
	//u<-u*w mod N
	Cuda_Mul_mod(temp_u,cy,temp_u,temp_w,N,temp_r,temp_r2);

	//w<-v+u mod N
	Cuda_Add_mod(temp_w,cy,temp_v,temp_u,N);
	//v<-v-u mod N
	Cuda_Sub_mod(temp_v,cy,temp_u,N);
	//w<-w^2 mod N
	Cuda_Square_mod(temp_w,cy,temp_w,N,temp_r,temp_r2);
	//v<-v^2 mod N
	Cuda_Square_mod(temp_v,cy,temp_v,N,temp_r,temp_r2);
	
	if (xplus==xminus) //same variable : in-place variant
	{
		Cuda_Mul_mod(zplus,cy,zminus,temp_w,N,temp_r,temp_r2);
		Cuda_Mul_mod(xplus,cy,xminus,temp_v,N,temp_r,temp_r2);
		//swap
		tmp=xplus[threadIdx.x];
		xplus[threadIdx.x]=zplus[threadIdx.x];
		zplus[threadIdx.x]=tmp;
	}
	else
	{
		//xplus <- zminus*u mod N
		Cuda_Mul_mod(xplus,cy,zminus,temp_w,N,temp_r,temp_r2);
		//zplus <- xminus*v mod N
		Cuda_Mul_mod(zplus,cy,xminus,temp_v,N,temp_r,temp_r2);
	}
}


#ifdef CC13
//Compute [3^power3*2^power2]A on the elliptic curve
//prepare for prac, ie set B and C equal to A and set A to [2]A
__global__ void Cuda_Ell_Mul_2_3(biguint_t *xAarg, biguint_t *zAarg, biguint_t *xBarg, biguint_t *zBarg, biguint_t *xCarg, biguint_t *zCarg, const biguint_t *darg, const unsigned int power2, const unsigned int power3)
{
	__shared__ unsigned int temp_r[2*SIZE_NUMBER];
	__shared__ unsigned int temp_r2[2*SIZE_NUMBER];
	__shared__ unsigned int temp_u[SIZE_NUMBER];
	__shared__ unsigned int temp_v[SIZE_NUMBER];
	__shared__ unsigned int temp_w[SIZE_NUMBER];

	__shared__ int cy[2*SIZE_NUMBER]; 
	
	__shared__ unsigned int xA[SIZE_NUMBER];
	__shared__ unsigned int zA[SIZE_NUMBER];
	__shared__ unsigned int xB[SIZE_NUMBER];
	__shared__ unsigned int zB[SIZE_NUMBER];
	
	__shared__ unsigned int d[SIZE_NUMBER];
	
	unsigned int i;

	//init
	xA[threadIdx.x]=xAarg[blockIdx.x][threadIdx.x];
	zA[threadIdx.x]=zAarg[blockIdx.x][threadIdx.x];
	xB[threadIdx.x]=xAarg[blockIdx.x][threadIdx.x];
	zB[threadIdx.x]=zAarg[blockIdx.x][threadIdx.x];

	d[threadIdx.x]=darg[blockIdx.x][threadIdx.x];

	temp_r[threadIdx.x]=0;	
	temp_r[SIZE_NUMBER + threadIdx.x]=0;	
	temp_r2[threadIdx.x]=0;	
	temp_r2[SIZE_NUMBER + threadIdx.x]=0;	
	temp_u[threadIdx.x]=0;	
	temp_v[threadIdx.x]=0;	
	temp_w[threadIdx.x]=0;	
	
	cy[threadIdx.x]=0;	
	cy[SIZE_NUMBER + threadIdx.x]=0;	

 	//__syncthreads();
	//Now one can begin the computation

	for(i=0;i<power2;i++)
	{
		Cuda_Ell_Dbl(xB, zB, xB, zB, Ncst, d, temp_u, temp_v, temp_r, temp_r2, cy);
	}
	
	for(i=0;i<power3;i++)
	{
		Cuda_Ell_Dbl(xA, zA, xB, zB, Ncst, d, temp_u, temp_v, temp_r, temp_r2, cy);
	Cuda_Ell_Add(xB, zB, xA, zA, xB, zB, xB, zB, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
	}

	Cuda_Ell_Dbl(xA, zA, xB, zB, Ncst, d, temp_u, temp_v, temp_r, temp_r2, cy);

	//End of the computation; Copy the results for the cpu
	xAarg[blockIdx.x][threadIdx.x]=xA[threadIdx.x];
	zAarg[blockIdx.x][threadIdx.x]=zA[threadIdx.x];
	xBarg[blockIdx.x][threadIdx.x]=xB[threadIdx.x];
	zBarg[blockIdx.x][threadIdx.x]=zB[threadIdx.x];
	xCarg[blockIdx.x][threadIdx.x]=xB[threadIdx.x];
	zCarg[blockIdx.x][threadIdx.x]=zB[threadIdx.x];
}

__global__ void Cuda_Ell_Add_Dbl(biguint_t *xAarg, biguint_t *zAarg, biguint_t *xBarg, biguint_t *zBarg, biguint_t *xCarg, biguint_t *zCarg, const biguint_t *darg)
{
	__shared__ unsigned int temp_r[2*SIZE_NUMBER];
	__shared__ unsigned int temp_r2[2*SIZE_NUMBER];
	__shared__ unsigned int temp_u[SIZE_NUMBER];
	__shared__ unsigned int temp_v[SIZE_NUMBER];
	__shared__ unsigned int temp_w[SIZE_NUMBER];

	__shared__ int cy[2*SIZE_NUMBER]; 
	
	__shared__ unsigned int xA[SIZE_NUMBER];
	__shared__ unsigned int zA[SIZE_NUMBER];
	__shared__ unsigned int xB[SIZE_NUMBER];
	__shared__ unsigned int zB[SIZE_NUMBER];
	__shared__ unsigned int xC[SIZE_NUMBER];
	__shared__ unsigned int zC[SIZE_NUMBER];
	
	__shared__ unsigned int d[SIZE_NUMBER];
	
	//init
	xA[threadIdx.x]=xAarg[blockIdx.x][threadIdx.x];
	zA[threadIdx.x]=zAarg[blockIdx.x][threadIdx.x];
	xB[threadIdx.x]=xBarg[blockIdx.x][threadIdx.x];
	zB[threadIdx.x]=zBarg[blockIdx.x][threadIdx.x];
	xC[threadIdx.x]=xCarg[blockIdx.x][threadIdx.x];
	zC[threadIdx.x]=zCarg[blockIdx.x][threadIdx.x];

	d[threadIdx.x]=darg[blockIdx.x][threadIdx.x];

	temp_r[threadIdx.x]=0;	
	temp_r[SIZE_NUMBER + threadIdx.x]=0;	
	temp_r2[threadIdx.x]=0;	
	temp_r2[SIZE_NUMBER + threadIdx.x]=0;	
	temp_u[threadIdx.x]=0;	
	temp_v[threadIdx.x]=0;	
	temp_w[threadIdx.x]=0;	
	
	cy[threadIdx.x]=0;	
	cy[SIZE_NUMBER + threadIdx.x]=0;	

 	//__syncthreads();
	//Now one can begin the computation

	//Add	
	Cuda_Ell_Add(xB, zB, xA, zA, xB, zB, xC, zC, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
	
	//Dbl
	Cuda_Ell_Dbl(xA, zA, xA, zA, Ncst, d, temp_u, temp_v, temp_r, temp_r2, cy);


	//End of the computation; Copy the results for the cpu
	xAarg[blockIdx.x][threadIdx.x]=xA[threadIdx.x];
	zAarg[blockIdx.x][threadIdx.x]=zA[threadIdx.x];
	xBarg[blockIdx.x][threadIdx.x]=xB[threadIdx.x];
	zBarg[blockIdx.x][threadIdx.x]=zB[threadIdx.x];
	xCarg[blockIdx.x][threadIdx.x]=xC[threadIdx.x];
	zCarg[blockIdx.x][threadIdx.x]=zC[threadIdx.x];
}

__global__ void Cuda_Ell_3Add(biguint_t *xAarg, biguint_t *zAarg, biguint_t *xBarg, biguint_t *zBarg, biguint_t *xCarg, biguint_t *zCarg)
{
	__shared__ unsigned int temp_r[2*SIZE_NUMBER];
	__shared__ unsigned int temp_r2[2*SIZE_NUMBER];
	__shared__ unsigned int temp_u[SIZE_NUMBER];
	__shared__ unsigned int temp_v[SIZE_NUMBER];
	__shared__ unsigned int temp_w[SIZE_NUMBER];

	__shared__ int cy[2*SIZE_NUMBER]; 
	
	__shared__ unsigned int xA[SIZE_NUMBER];
	__shared__ unsigned int zA[SIZE_NUMBER];
	__shared__ unsigned int xB[SIZE_NUMBER];
	__shared__ unsigned int zB[SIZE_NUMBER];
	__shared__ unsigned int xC[SIZE_NUMBER];
	__shared__ unsigned int zC[SIZE_NUMBER];
	__shared__ unsigned int xT[SIZE_NUMBER];
	__shared__ unsigned int zT[SIZE_NUMBER];
	__shared__ unsigned int xT2[SIZE_NUMBER];
	__shared__ unsigned int zT2[SIZE_NUMBER];
	
	//init
	xA[threadIdx.x]=xAarg[blockIdx.x][threadIdx.x];
	zA[threadIdx.x]=zAarg[blockIdx.x][threadIdx.x];
	xB[threadIdx.x]=xBarg[blockIdx.x][threadIdx.x];
	zB[threadIdx.x]=zBarg[blockIdx.x][threadIdx.x];
	xC[threadIdx.x]=xCarg[blockIdx.x][threadIdx.x];
	zC[threadIdx.x]=zCarg[blockIdx.x][threadIdx.x];

	xT[threadIdx.x]=0;
	zT[threadIdx.x]=0;
	xT2[threadIdx.x]=0;
	zT2[threadIdx.x]=0;
	
	temp_r[threadIdx.x]=0;	
	temp_r[SIZE_NUMBER + threadIdx.x]=0;	
	temp_r2[threadIdx.x]=0;	
	temp_r2[SIZE_NUMBER + threadIdx.x]=0;	
	temp_u[threadIdx.x]=0;	
	temp_v[threadIdx.x]=0;	
	temp_w[threadIdx.x]=0;	
	
	cy[threadIdx.x]=0;	
	cy[SIZE_NUMBER + threadIdx.x]=0;	

 	//__syncthreads();
	//Now one can begin the computation

	//Add	
	Cuda_Ell_Add(xT, zT, xA, zA, xB, zB, xC, zC, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
	Cuda_Ell_Add(xT2, zT2, xT, zT, xA, zA, xB, zB, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
	Cuda_Ell_Add(xB, zB, xB, zB, xT, zT, xA, zA, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
	
	//End of the computation; Copy the results for the cpu
	xAarg[blockIdx.x][threadIdx.x]=xT2[threadIdx.x];
	zAarg[blockIdx.x][threadIdx.x]=zT2[threadIdx.x];
	xBarg[blockIdx.x][threadIdx.x]=xB[threadIdx.x];
	zBarg[blockIdx.x][threadIdx.x]=zB[threadIdx.x];
}

__global__ void Cuda_Ell_Add_Perm(biguint_t *xAarg, biguint_t *zAarg, biguint_t *xBarg, biguint_t *zBarg, biguint_t *xCarg, biguint_t *zCarg)
{
	__shared__ unsigned int temp_r[2*SIZE_NUMBER];
	__shared__ unsigned int temp_r2[2*SIZE_NUMBER];
	__shared__ unsigned int temp_u[SIZE_NUMBER];
	__shared__ unsigned int temp_v[SIZE_NUMBER];
	__shared__ unsigned int temp_w[SIZE_NUMBER];

	__shared__ int cy[2*SIZE_NUMBER]; 
	
	__shared__ unsigned int xA[SIZE_NUMBER];
	__shared__ unsigned int zA[SIZE_NUMBER];
	__shared__ unsigned int xB[SIZE_NUMBER];
	__shared__ unsigned int zB[SIZE_NUMBER];
	__shared__ unsigned int xC[SIZE_NUMBER];
	__shared__ unsigned int zC[SIZE_NUMBER];
	
	//init
	xA[threadIdx.x]=xAarg[blockIdx.x][threadIdx.x];
	zA[threadIdx.x]=zAarg[blockIdx.x][threadIdx.x];
	xB[threadIdx.x]=xBarg[blockIdx.x][threadIdx.x];
	zB[threadIdx.x]=zBarg[blockIdx.x][threadIdx.x];
	xC[threadIdx.x]=xCarg[blockIdx.x][threadIdx.x];
	zC[threadIdx.x]=zCarg[blockIdx.x][threadIdx.x];

	temp_r[threadIdx.x]=0;	
	temp_r[SIZE_NUMBER + threadIdx.x]=0;	
	temp_r2[threadIdx.x]=0;	
	temp_r2[SIZE_NUMBER + threadIdx.x]=0;	
	temp_u[threadIdx.x]=0;	
	temp_v[threadIdx.x]=0;	
	temp_w[threadIdx.x]=0;	
	
	cy[threadIdx.x]=0;	
	cy[SIZE_NUMBER + threadIdx.x]=0;	

 	//__syncthreads();
	//Now one can begin the computation

	//Add	
	Cuda_Ell_Add(xA, zA, xB, zB, xA, zA, xC, zC, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
	
	//End of the computation; Copy the results for the cpu
	xBarg[blockIdx.x][threadIdx.x]=xA[threadIdx.x];
	zBarg[blockIdx.x][threadIdx.x]=zA[threadIdx.x];
	xCarg[blockIdx.x][threadIdx.x]=xB[threadIdx.x];
	zCarg[blockIdx.x][threadIdx.x]=zB[threadIdx.x];
}

__global__ void Cuda_Ell_Dbl_3Add(biguint_t *xAarg, biguint_t *zAarg, biguint_t *xBarg, biguint_t *zBarg, biguint_t *xCarg, biguint_t *zCarg, const biguint_t *darg)
{
	__shared__ unsigned int temp_r[2*SIZE_NUMBER];
	__shared__ unsigned int temp_r2[2*SIZE_NUMBER];
	__shared__ unsigned int temp_u[SIZE_NUMBER];
	__shared__ unsigned int temp_v[SIZE_NUMBER];
	__shared__ unsigned int temp_w[SIZE_NUMBER];

	__shared__ int cy[2*SIZE_NUMBER]; 
	
	__shared__ unsigned int xA[SIZE_NUMBER];
	__shared__ unsigned int zA[SIZE_NUMBER];
	__shared__ unsigned int xB[SIZE_NUMBER];
	__shared__ unsigned int zB[SIZE_NUMBER];
	__shared__ unsigned int xC[SIZE_NUMBER];
	__shared__ unsigned int zC[SIZE_NUMBER];
	__shared__ unsigned int xT[SIZE_NUMBER];
	__shared__ unsigned int zT[SIZE_NUMBER];
	__shared__ unsigned int xT2[SIZE_NUMBER];
	__shared__ unsigned int zT2[SIZE_NUMBER];

	__shared__ unsigned int d[SIZE_NUMBER];
	
	//init
	xA[threadIdx.x]=xAarg[blockIdx.x][threadIdx.x];
	zA[threadIdx.x]=zAarg[blockIdx.x][threadIdx.x];
	xB[threadIdx.x]=xBarg[blockIdx.x][threadIdx.x];
	zB[threadIdx.x]=zBarg[blockIdx.x][threadIdx.x];
	xC[threadIdx.x]=xCarg[blockIdx.x][threadIdx.x];
	zC[threadIdx.x]=zCarg[blockIdx.x][threadIdx.x];

	xT[threadIdx.x]=0;
	zT[threadIdx.x]=0;
	xT2[threadIdx.x]=0;
	zT2[threadIdx.x]=0;

	d[threadIdx.x]=darg[blockIdx.x][threadIdx.x];

	temp_r[threadIdx.x]=0;	
	temp_r[SIZE_NUMBER + threadIdx.x]=0;	
	temp_r2[threadIdx.x]=0;	
	temp_r2[SIZE_NUMBER + threadIdx.x]=0;	
	temp_u[threadIdx.x]=0;	
	temp_v[threadIdx.x]=0;	
	temp_w[threadIdx.x]=0;	
	
	cy[threadIdx.x]=0;	
	cy[SIZE_NUMBER + threadIdx.x]=0;	

 	//__syncthreads();
	//Now one can begin the computation
	Cuda_Ell_Dbl(xT, zT, xA, zA, Ncst, d, temp_u, temp_v, temp_r, temp_r2, cy);
	Cuda_Ell_Add(xT2, zT2, xA, zA, xB, zB, xC, zC, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
	Cuda_Ell_Add(xA, zA, xT, zT, xA, zA, xA, zA, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
	Cuda_Ell_Add(xT, zT, xT, zT, xT2, zT2, xC, zC, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
	
	//End of the computation; Copy the results for the cpu
	xAarg[blockIdx.x][threadIdx.x]=xA[threadIdx.x];
	zAarg[blockIdx.x][threadIdx.x]=zA[threadIdx.x];
	xBarg[blockIdx.x][threadIdx.x]=xT[threadIdx.x];
	zBarg[blockIdx.x][threadIdx.x]=zT[threadIdx.x];
	xCarg[blockIdx.x][threadIdx.x]=xB[threadIdx.x];
	zCarg[blockIdx.x][threadIdx.x]=zB[threadIdx.x];
}

__global__ void Cuda_Ell_2Add_Dbl_Add(biguint_t *xAarg, biguint_t *zAarg, biguint_t *xBarg, biguint_t *zBarg, biguint_t *xCarg, biguint_t *zCarg, const biguint_t *darg, const int version)
{
	__shared__ unsigned int temp_r[2*SIZE_NUMBER];
	__shared__ unsigned int temp_r2[2*SIZE_NUMBER];
	__shared__ unsigned int temp_u[SIZE_NUMBER];
	__shared__ unsigned int temp_v[SIZE_NUMBER];
	__shared__ unsigned int temp_w[SIZE_NUMBER];

	__shared__ int cy[2*SIZE_NUMBER]; 
	
	__shared__ unsigned int xA[SIZE_NUMBER];
	__shared__ unsigned int zA[SIZE_NUMBER];
	__shared__ unsigned int xB[SIZE_NUMBER];
	__shared__ unsigned int zB[SIZE_NUMBER];
	__shared__ unsigned int xC[SIZE_NUMBER];
	__shared__ unsigned int zC[SIZE_NUMBER];
	__shared__ unsigned int xT[SIZE_NUMBER];
	__shared__ unsigned int zT[SIZE_NUMBER];
	
	__shared__ unsigned int d[SIZE_NUMBER];
	
	//init
	xA[threadIdx.x]=xAarg[blockIdx.x][threadIdx.x];
	zA[threadIdx.x]=zAarg[blockIdx.x][threadIdx.x];
	xB[threadIdx.x]=xBarg[blockIdx.x][threadIdx.x];
	zB[threadIdx.x]=zBarg[blockIdx.x][threadIdx.x];
	xC[threadIdx.x]=xCarg[blockIdx.x][threadIdx.x];
	zC[threadIdx.x]=zCarg[blockIdx.x][threadIdx.x];

	xT[threadIdx.x]=0;
	zT[threadIdx.x]=0;

	d[threadIdx.x]=darg[blockIdx.x][threadIdx.x];

	temp_r[threadIdx.x]=0;	
	temp_r[SIZE_NUMBER + threadIdx.x]=0;	
	temp_r2[threadIdx.x]=0;	
	temp_r2[SIZE_NUMBER + threadIdx.x]=0;	
	temp_u[threadIdx.x]=0;	
	temp_v[threadIdx.x]=0;	
	temp_w[threadIdx.x]=0;	
	
	cy[threadIdx.x]=0;	
	cy[SIZE_NUMBER + threadIdx.x]=0;	

 	//__syncthreads();
	//Now one can begin the computation
	Cuda_Ell_Add(xT, zT, xA, zA, xB, zB, xC, zC, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
	
	if (version==1)
		Cuda_Ell_Add(xC, zC, xT, zT, xA, zA, xB, zB, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
	else 
	{
		Cuda_Ell_Add(xC, zC, xC, zC, xA, zA, xB, zB, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
		xB[threadIdx.x]=xT[threadIdx.x];
		zB[threadIdx.x]=zT[threadIdx.x];
	}
	Cuda_Ell_Dbl(xT, zT, xA, zA, Ncst, d, temp_u, temp_v, temp_r, temp_r2, cy);
	Cuda_Ell_Add(xA, zA, xA, zA, xT, zT, xA, zA, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
	
	//End of the computation; Copy the results for the cpu
	xAarg[blockIdx.x][threadIdx.x]=xA[threadIdx.x];
	zAarg[blockIdx.x][threadIdx.x]=zA[threadIdx.x];
	if (version==1)
	{
		xBarg[blockIdx.x][threadIdx.x]=xC[threadIdx.x];
		zBarg[blockIdx.x][threadIdx.x]=zC[threadIdx.x];
	}
	else
	{
		xBarg[blockIdx.x][threadIdx.x]=xB[threadIdx.x];
		zBarg[blockIdx.x][threadIdx.x]=zB[threadIdx.x];
		xCarg[blockIdx.x][threadIdx.x]=xC[threadIdx.x];
		zCarg[blockIdx.x][threadIdx.x]=zC[threadIdx.x];
		
	}

}

__global__ void Cuda_Ell_Final(biguint_t *xAarg, biguint_t *zAarg, biguint_t *xBarg, biguint_t *zBarg, biguint_t *xCarg, biguint_t *zCarg, const biguint_t *darg)
{
	__shared__ unsigned int temp_r[2*SIZE_NUMBER];
	__shared__ unsigned int temp_r2[2*SIZE_NUMBER];
	__shared__ unsigned int temp_u[SIZE_NUMBER];
	__shared__ unsigned int temp_v[SIZE_NUMBER];
	__shared__ unsigned int temp_w[SIZE_NUMBER];

	__shared__ int cy[2*SIZE_NUMBER]; 
	
	__shared__ unsigned int xA[SIZE_NUMBER];
	__shared__ unsigned int zA[SIZE_NUMBER];
	__shared__ unsigned int xB[SIZE_NUMBER];
	__shared__ unsigned int zB[SIZE_NUMBER];
	__shared__ unsigned int xC[SIZE_NUMBER];
	__shared__ unsigned int zC[SIZE_NUMBER];
	
	__shared__ unsigned int d[SIZE_NUMBER];
	
	//init
	xA[threadIdx.x]=xAarg[blockIdx.x][threadIdx.x];
	zA[threadIdx.x]=zAarg[blockIdx.x][threadIdx.x];
	xB[threadIdx.x]=xBarg[blockIdx.x][threadIdx.x];
	zB[threadIdx.x]=zBarg[blockIdx.x][threadIdx.x];
	xC[threadIdx.x]=xCarg[blockIdx.x][threadIdx.x];
	zC[threadIdx.x]=zCarg[blockIdx.x][threadIdx.x];

	d[threadIdx.x]=darg[blockIdx.x][threadIdx.x];

	temp_r[threadIdx.x]=0;	
	temp_r[SIZE_NUMBER + threadIdx.x]=0;	
	temp_r2[threadIdx.x]=0;	
	temp_r2[SIZE_NUMBER + threadIdx.x]=0;	
	temp_u[threadIdx.x]=0;	
	temp_v[threadIdx.x]=0;	
	temp_w[threadIdx.x]=0;	
	
	cy[threadIdx.x]=0;	
	cy[SIZE_NUMBER + threadIdx.x]=0;	

 	//__syncthreads();
	//Now one can begin the computation

	
	Cuda_Ell_Add(xB, zB, xA, zA, xB, zB, xC, zC, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
	Cuda_Ell_Dbl(xA, zA, xB, zB, Ncst, d, temp_u, temp_v, temp_r, temp_r2, cy);

	//End of the computation; Copy the results for the cpu
	xAarg[blockIdx.x][threadIdx.x]=xA[threadIdx.x];
	zAarg[blockIdx.x][threadIdx.x]=zA[threadIdx.x];
	xBarg[blockIdx.x][threadIdx.x]=xB[threadIdx.x];
	zBarg[blockIdx.x][threadIdx.x]=zB[threadIdx.x];
	xCarg[blockIdx.x][threadIdx.x]=xB[threadIdx.x];
	zCarg[blockIdx.x][threadIdx.x]=zB[threadIdx.x];
	
}

#endif

#ifdef CC20
__device__ void Cuda_Swap(biguint_t A, biguint_t B)
{
	unsigned int temp=A[threadIdx.x];
	A[threadIdx.x]=B[threadIdx.x];
	B[threadIdx.x]=temp;
}

__device__ void Cuda_Circular_Perm(biguint_t A, biguint_t B, biguint_t C)
{
	unsigned int temp=A[threadIdx.x];
	A[threadIdx.x]=B[threadIdx.x];
	B[threadIdx.x]=C[threadIdx.x];
	C[threadIdx.x]=temp;
}
//Compute [3^power3*2^power2]A on the elliptic curve
//prepare for prac, ie set B and C equal to A and set A to [2]A
__global__ void Cuda_Ell_Mul_2_3(biguint_t *xAarg, biguint_t *zAarg, const biguint_t *darg, const unsigned int power2, const unsigned int power3)
{
	__shared__ unsigned int temp_r[2*SIZE_NUMBER];
	__shared__ unsigned int temp_r2[2*SIZE_NUMBER];
	__shared__ unsigned int temp_u[SIZE_NUMBER];
	__shared__ unsigned int temp_v[SIZE_NUMBER];
	__shared__ unsigned int temp_w[SIZE_NUMBER];

	__shared__ int cy[2*SIZE_NUMBER]; 
	
	__shared__ unsigned int xA[SIZE_NUMBER];
	__shared__ unsigned int zA[SIZE_NUMBER];
	__shared__ unsigned int xT[SIZE_NUMBER];
	__shared__ unsigned int zT[SIZE_NUMBER];
	
	__shared__ unsigned int dc[SIZE_NUMBER];
	
	unsigned int i;

	//init
	xA[threadIdx.x]=xAarg[blockIdx.x][threadIdx.x];
	zA[threadIdx.x]=zAarg[blockIdx.x][threadIdx.x];
	xT[threadIdx.x]=0;
	zT[threadIdx.x]=0;

	dc[threadIdx.x]=darg[blockIdx.x][threadIdx.x];

	temp_r[threadIdx.x]=0;	
	temp_r[SIZE_NUMBER + threadIdx.x]=0;	
	temp_r2[threadIdx.x]=0;	
	temp_r2[SIZE_NUMBER + threadIdx.x]=0;	
	temp_u[threadIdx.x]=0;	
	temp_v[threadIdx.x]=0;	
	temp_w[threadIdx.x]=0;	
	
	cy[threadIdx.x]=0;	
	cy[SIZE_NUMBER + threadIdx.x]=0;	

	//Now one can begin the computation

	//printf("power2=%u power3=%u\n",power2,power3);

	for(i=0;i<power2;i++)
	{
		Cuda_Ell_Dbl(xA, zA, xA, zA, Ncst, dc, temp_u, temp_v, temp_r, temp_r2, cy);
	}
	
	for(i=0;i<power3;i++)
	{
		Cuda_Ell_Dbl(xT, zT, xA, zA, dc, Ncst, temp_u, temp_v, temp_r, temp_r2, cy);
		Cuda_Ell_Add(xA, zA, xT, zT, xA, zA, xA, zA, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
	}

	if (threadIdx.x==0 && blockIdx.x==0)
		printf("%u+%u*2^32+%u*2^64+...\n",invmodcst[0],invmodcst[1],invmodcst[2]);
	if (threadIdx.x==0 && blockIdx.x==0)
		printf("%u+%u*2^32+%u*2^64+...\n",dc[0],dc[1],dc[2]);
	if (threadIdx.x==0 && blockIdx.x==0)
		printf("%u+%u*2^32+%u*2^64+...\n",Ncst[0],Ncst[1],Ncst[2]);
	if (threadIdx.x==0 && blockIdx.x==0)
		printf("%u+%u*2^32+%u*2^64+...\n",zAarg[0],zAarg[1],zAarg[2]);
	if (threadIdx.x==0 && blockIdx.x==0)
		printf("%u+%u*2^32+%u*2^64+...\n",zA[0],zA[1],zA[2]);
	//if (threadIdx.x==0)
	//	printf("power2=%u power3=%u\n",power2,power3);
	//End of the computation; Copy the results for the cpu
	xAarg[blockIdx.x][threadIdx.x]=xA[threadIdx.x];
	zAarg[blockIdx.x][threadIdx.x]=zA[threadIdx.x];
}

__global__ void Cuda_PRAC(biguint_t *xAarg, biguint_t *zAarg, const biguint_t *darg, unsigned int PI, unsigned int B1, double val)
{
	__shared__ unsigned int temp_r[2*SIZE_NUMBER];
	__shared__ unsigned int temp_r2[2*SIZE_NUMBER];
	__shared__ unsigned int temp_u[SIZE_NUMBER];
	__shared__ unsigned int temp_v[SIZE_NUMBER];
	__shared__ unsigned int temp_w[SIZE_NUMBER];

	__shared__ int cy[2*SIZE_NUMBER]; 
	
	__shared__ unsigned int xA[SIZE_NUMBER];
	__shared__ unsigned int zA[SIZE_NUMBER];
	__shared__ unsigned int xB[SIZE_NUMBER];
	__shared__ unsigned int zB[SIZE_NUMBER];
	__shared__ unsigned int xC[SIZE_NUMBER];
	__shared__ unsigned int zC[SIZE_NUMBER];
	__shared__ unsigned int xT[SIZE_NUMBER];
	__shared__ unsigned int zT[SIZE_NUMBER];
	__shared__ unsigned int xT2[SIZE_NUMBER];
	__shared__ unsigned int zT2[SIZE_NUMBER];

	__shared__ unsigned int dc[SIZE_NUMBER];
	
	unsigned int e;
	unsigned int d;
	unsigned int r;
	unsigned int pp=PI;

	//init
	xA[threadIdx.x]=xAarg[blockIdx.x][threadIdx.x];
	zA[threadIdx.x]=zAarg[blockIdx.x][threadIdx.x];
	xB[threadIdx.x]=xA[threadIdx.x];
	zB[threadIdx.x]=zA[threadIdx.x];
	xC[threadIdx.x]=xA[threadIdx.x];
	zC[threadIdx.x]=zA[threadIdx.x];
	xT[threadIdx.x]=0;
	zT[threadIdx.x]=0;
	xT2[threadIdx.x]=0;
	zT2[threadIdx.x]=0;

	dc[threadIdx.x]=darg[blockIdx.x][threadIdx.x];

	temp_r[threadIdx.x]=0;	
	temp_r[SIZE_NUMBER + threadIdx.x]=0;	
	temp_r2[threadIdx.x]=0;	
	temp_r2[SIZE_NUMBER + threadIdx.x]=0;	
	temp_u[threadIdx.x]=0;	
	temp_v[threadIdx.x]=0;	
	temp_w[threadIdx.x]=0;	
	
	cy[threadIdx.x]=0;	
	cy[SIZE_NUMBER + threadIdx.x]=0;	

 	//__syncthreads();
	//Now one can begin the computation

	while (pp<=B1)
	{
  	d = PI;
  	r = (unsigned int) ((double) d * val + 0.5);
  	d = PI - r;
  	e = 2 * r - PI;

	
		Cuda_Ell_Dbl(xA, zA, xA, zA, Ncst, dc, temp_u, temp_v, temp_r, temp_r2, cy);
		
		while (d != e)
	  {
	 		if (d < e)
	  	{
		  	r = d;
	    	d = e;
	   		e = r;
				Cuda_Swap(xA,xB);
				Cuda_Swap(zA,zB);
	  	}
	      
			// do the first line of Table 4 whose condition qualifies 
	  	if (d - e <= e / 4 && ((d + e) % 3) == 0)
	  	{ // condition 1 
	  		d = (2 * d - e) / 3;
	    	e = (e - d) / 2;
				Cuda_Ell_Add(xT, zT, xA, zA, xB, zB, xC, zC, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
				Cuda_Ell_Add(xT2, zT2, xT, zT, xA, zA, xB, zB, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
				Cuda_Ell_Add(xB, zB, xB, zB, xT, zT, xA, zA, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
				Cuda_Swap(xA,xT2);
				Cuda_Swap(zA,zT2);
			}
	
	  	else if (d - e <= e / 4 && (d - e) % 6 == 0)
	  	{ // condition 2 
	  		d = (d - e) / 2;
				Cuda_Ell_Add(xB, zB, xA, zA, xB, zB, xC, zC, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
				Cuda_Ell_Dbl(xA, zA, xA, zA, Ncst, dc, temp_u, temp_v, temp_r, temp_r2, cy);
	  	}
	
	  	else if ((d + 3) / 4 <= e)
	  	{ // condition 3 
	  		d -= e;
				Cuda_Ell_Add(xT, zT, xB, zB, xA, zA, xC, zC, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
				//circular permutation
				Cuda_Circular_Perm(xB,xT,xC);
				Cuda_Circular_Perm(zB,zT,zC);
			}
	
	  	else if ((d + e) % 2 == 0)
			{ // condition 4 
	  		d = (d - e) / 2;
				Cuda_Ell_Add(xB, zB, xB, zB, xA, zA, xC, zC, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
				Cuda_Ell_Dbl(xA, zA, xA, zA, Ncst, dc, temp_u, temp_v, temp_r, temp_r2, cy);
			}
	
	  	// now d+e is odd 
	  	else if (d % 2 == 0)
			{ // condition 5 
	  		d /= 2;
				Cuda_Ell_Add(xC, zC, xC, zC, xA, zA, xB, zB, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
				Cuda_Ell_Dbl(xA, zA, xA, zA, Ncst, dc, temp_u, temp_v, temp_r, temp_r2, cy);
			}
	
	  	// now d is odd, e is even 
	  	else if (d % 3 == 0)
			{ // condition 6 
	  		d = d / 3 - e;
				Cuda_Ell_Dbl(xT, zT, xA, zA, Ncst, dc, temp_u, temp_v, temp_r, temp_r2, cy);
				Cuda_Ell_Add(xT2, zT2, xA, zA, xB, zB, xC, zC, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
				Cuda_Ell_Add(xA, zA, xT, zT, xA, zA, xA, zA, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
				Cuda_Ell_Add(xT, zT, xT, zT, xT2, zT2, xC, zC, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
				//circular permutation
				Cuda_Circular_Perm(xC,xB,xT);
				Cuda_Circular_Perm(zC,zB,zT);
			}
	
	  	else if ((d + e) % 3 == 0)
			{ // condition 7 
	  		d = (d - 2 * e) / 3;
				Cuda_Ell_Add(xT, zT, xA, zA, xB, zB, xC, zC, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
				Cuda_Ell_Add(xB, zB, xT, zT, xA, zA, xB, zB, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
				Cuda_Ell_Dbl(xT, zT, xA, zA, Ncst, dc, temp_u, temp_v, temp_r, temp_r2, cy);
				Cuda_Ell_Add(xA, zA, xA, zA, xT, zT, xA, zA, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
			}
	
	  
			else if ((d - e) % 3 == 0)
			{ // condition 8 
	  		d = (d - e) / 3;
				Cuda_Ell_Add(xT, zT, xA, zA, xB, zB, xC, zC, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
				Cuda_Ell_Add(xC, zC, xC, zC, xA, zA, xB, zB, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
				Cuda_Swap(xB,xT);
				Cuda_Swap(zB,zT);
				Cuda_Ell_Dbl(xT, zT, xA, zA, Ncst, dc, temp_u, temp_v, temp_r, temp_r2, cy);
				Cuda_Ell_Add(xA, zA, xA, zA, xT, zT, xA, zA, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
			}
						
			else // necessarily e is even here 
			{ // condition 9 
	  		e /= 2;
				Cuda_Ell_Add(xC, zC, xC, zC, xB, zB, xA, zA, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);
				Cuda_Ell_Dbl(xB, zB, xB, zB, Ncst, dc, temp_u, temp_v, temp_r, temp_r2, cy);
			}
		}

		Cuda_Ell_Add(xA, zA, xA, zA, xB, zB, xC, zC, Ncst, temp_u, temp_v, temp_w, temp_r, temp_r2, cy);

	pp*=PI;
	}


//End of the computation; Copy the results for the cpu
	xAarg[blockIdx.x][threadIdx.x]=xA[threadIdx.x];
	zAarg[blockIdx.x][threadIdx.x]=zA[threadIdx.x];
}
#endif

__host__ void cuda_copy_cst(biguint_t h_N, biguint_t h_invmod)
{
	hipMemcpyToSymbol(HIP_SYMBOL(invmodcst),h_invmod,sizeof(biguint_t));
	hipMemcpyToSymbol(HIP_SYMBOL(Ncst),h_N,sizeof(biguint_t));
}
