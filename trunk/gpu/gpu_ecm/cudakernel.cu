#include "hip/hip_runtime.h"
#include "def.h"
#include "cudakernel.h"

__constant__ __device__ digit_t d_invNcst;
__device__ biguint_t d_Ncst;
__device__ biguint_t d_3Ncst;
__device__ biguint_t d_Mcst;


#define errCheck(err) cuda_errCheck (err, __FILE__, __LINE__)
#define hipMalloc(d, size) errCheck (hipMalloc (d, size))
#define cudaMemcpyHtoD(d, h, size) errCheck (hipMemcpy ((void *) d, \
                                    (void *) h, size, hipMemcpyHostToDevice))
#define cudaMemcpyDtoH(h, d, size) errCheck (hipMemcpy ((void *) h, \
                                    (void *) d, size, hipMemcpyDeviceToHost))


/******************************/
/* Host code handling the GPU */
/******************************/

inline void cuda_errCheck (hipError_t err, const char *file, const int line)
{
  if( err != hipSuccess ) 
  {
    fprintf(stderr, "%s(%i) : Error cuda : %s.\n",
              file, line, hipGetErrorString( err) );
    exit(EXIT_FAILURE);
  }
}


extern "C" 
int select_GPU (int device, int number_of_curves, FILE *OUTPUT_VERBOSE)
{
  hipDeviceProp_t deviceProp;
  hipError_t err;
        
  fprintf(OUTPUT_VERBOSE, "#Compiled for a NVIDIA GPU with " 
          "compute capability %d.%d.\n", MAJOR, MINOR);

  if (device!=-1)
  {
    fprintf(OUTPUT_VERBOSE,"#Device %d is required.\n",device);

    err= hipSetDevice(device);
    if (err != hipSuccess)
    {
      fprintf(stderr, "Error: Could not use device %d\n",device);
      fprintf(stderr, "Error msg: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  }
  
  err = hipGetDevice (&device);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Error: no active device\n");
    fprintf(stderr, "Error msg: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipGetDeviceProperties (&deviceProp, device);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Error while getting device's properties\n");
    fprintf(stderr, "Error msg: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  int minor = deviceProp.minor;
  int major = deviceProp.major;
  int MPcount = deviceProp.multiProcessorCount;

  if (10 * major + minor < 10 * MAJOR + MINOR)
  {
    fprintf(stderr, "Error: Device %d have a compute capability of %d.%d " 
                    "(required %d.%d).\n", device, major, minor, MAJOR, MINOR);
    exit(EXIT_FAILURE);
  }

  fprintf(OUTPUT_VERBOSE, "#Will use device %d : %s, compute capability %d.%d, "
          "%d MPs.\n", device, deviceProp.name, major, minor, MPcount);


  hipSetDeviceFlags(hipDeviceScheduleAuto); 
  //hipSetDeviceFlags(hipDeviceScheduleYield); 
  //hipSetDeviceFlags(hipDeviceScheduleSpin); //the other make performance
  //worse

  //number_of_curves should be a multiple of CURVES_BY_BLOCK
  number_of_curves=(number_of_curves/CURVES_BY_BLOCK)*CURVES_BY_BLOCK;
  if (number_of_curves==0)
    number_of_curves = MPcount * CURVES_BY_MP;

  return number_of_curves;
}

extern "C"
void cuda_Main (biguint_t h_N, biguint_t h_3N, biguint_t h_M, digit_t h_invN, 
                    biguint_t *h_xarray, biguint_t *h_zarray, 
                    biguint_t *h_x2array, biguint_t *h_z2array, mpz_t s,
                    unsigned int firstinvd, unsigned int number_of_curves, 
                    FILE *OUTPUT_VERBOSE, FILE *OUTPUT_VVERBOSE) 
{ 
  size_t j;
  biguint_t *d_xA, *d_zA, *d_xB, *d_zB;

  size_t array_size = sizeof(biguint_t) * number_of_curves;

  dim3 dimBlock (NB_DIGITS, CURVES_BY_BLOCK);
  dim3 dimGrid (number_of_curves/CURVES_BY_BLOCK);

  fprintf(OUTPUT_VVERBOSE, "Block: %ux%ux%u Grid: %ux%ux%u\n", dimBlock.x, 
                      dimBlock.y, dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);

  hipMalloc (&d_xA, array_size);
  hipMalloc (&d_zA, array_size);
  hipMalloc (&d_xB, array_size);
  hipMalloc (&d_zB, array_size);

  /* Copy into the gpu memory */
  hipMemcpyToSymbol(HIP_SYMBOL(d_invNcst), (void *) &h_invN, sizeof(digit_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_Ncst), (void *) h_N, sizeof(biguint_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_3Ncst), (void *) h_3N, sizeof(biguint_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_Mcst), (void *) h_M, sizeof(biguint_t));

  cudaMemcpyHtoD (d_xA, h_xarray, array_size);
  cudaMemcpyHtoD (d_zA, h_zarray, array_size);
  cudaMemcpyHtoD (d_xB, h_x2array, array_size);
  cudaMemcpyHtoD (d_zB, h_z2array, array_size);

  /* Double-and-add loop: it calls the GPU for each bits of s */
  for (j = mpz_sizeinbase (s, 2) - 1; j>0; j-- )
  {
    if (mpz_tstbit (s, j-1) == 1)
      Cuda_Ell_DblAdd<<<dimGrid,dimBlock>>>(d_xB, d_zB, d_xA, d_zA, firstinvd);
    else
      Cuda_Ell_DblAdd<<<dimGrid,dimBlock>>>(d_xA, d_zA, d_xB, d_zB, firstinvd);

    //maybe only for debug mode??
    errCheck (hipGetLastError()); 
  }

  /* Get the results back from device memory */
  cudaMemcpyDtoH (h_xarray, d_xA, array_size);
  cudaMemcpyDtoH (h_zarray, d_zA, array_size);

  hipFree ((void *) d_xA);
  hipFree ((void *) d_zA);
  hipFree ((void *) d_xB);
  hipFree ((void *) d_zB);
}



/***************/
/* Device code */
/***************/


#define __add_cc(r,a,b) __asm__("add.cc.u32 %0,%1, %2;" :"=r"(r):"r"(a),"r"(b)) 
#define __addc_cc(r,a,b) __asm__("addc.cc.u32 %0,%1, %2;":"=r"(r):"r"(a),"r"(b)) 

#define __sub_cc(r,a,b) __asm__("sub.cc.u32 %0,%1, %2;" :"=r"(r):"r"(a),"r"(b)) 

#define __addcy(carry) __asm__("addc.s32 %0, 0, 0;" :"=r"(carry)) 
#define __addcy2(carry) __asm__("addc.s32 %0, %0, 0;" :"+r"(carry)) 

#define __subcy(carry) __asm__("subc.s32 %0, 0, 0;" :"=r"(carry)) 
#define __subcy2(carry) __asm__("subc.s32 %0, %0, 0;" :"+r"(carry)) 

#define __mul(h,l,a,b) __asm__("mul.hi.u32 %0,%2,%3;\n\t" "mul.lo.u32 %1,%2,%3;"\
                                            : "=r"(h), "=r"(l) : "r"(a), "r"(b))

#define __mad_lo(r,a,b,c) __asm__("mad.lo.u32 %0,%1,%2,%3;" \
                                            : "=r"(r) : "r"(a), "r"(b), "r"(c))
#define __mad_hi(r,a,b,c) __asm__("mad.hi.u32 %0,%1,%2,%3;" \
                                            : "=r"(r) : "r"(a), "r"(b), "r"(c))




//  (A > B)?, returns 1(true), -1(false) or 0(a=b) 
//Assume A and B are normalize (no carry or borrow)
__device__ int Cuda_Cmp(const biguint_t A, const biguint_t B)
{
  int i;
  for (i = NB_DIGITS-1;i>=0;i--)
  {
    if (A[i] > B[i])
      return 1;
    else if (A[i] < B[i])
      return -1;
  }
  return 0;
}

//Assume cy[threadIdx.x] = 0,+/-1
__device__ void Cuda_Normalize(biguint_t A,dbigint_t cy)
{
  carry_t cytemp;
  cytemp = cy[threadIdx.x];
  cy[threadIdx.x]=0;
  int tmp=threadIdx.x+1 % NB_DIGITS;

  if (cytemp==1)
  {
    A[tmp]++;
    if (A[tmp]==0)
      cy[tmp]=cytemp;
  }
  else if (cytemp==-1) 
  {
    if (A[tmp]==0)
      cy[tmp]=cytemp;
    A[tmp]--;
  } 
}

__device__ void Cuda_Fully_Normalize(biguint_t A,dbigint_t cy)
{
  
  do
  {
    Cuda_Normalize(A,cy);
  }while(__any(cy[threadIdx.x])!=0);
  
}

__device__ void Cuda_Add 
(biguint_t r, dbigint_t cy ,const biguint_t a, const biguint_t b)
{
  __add_cc(r[threadIdx.x],a[threadIdx.x],b[threadIdx.x]);
  __addcy(cy[threadIdx.x]);
}

__device__ void Cuda_Subc 
(biguint_t r, dbigint_t cy, const biguint_t a, const biguint_t b)
{
  __sub_cc(r[threadIdx.x],a[threadIdx.x],b[threadIdx.x]);
  __subcy2(cy[threadIdx.x]);
}

__device__ void Cuda_Sub 
(biguint_t r, dbigint_t cy, const biguint_t a, const biguint_t b)
{
  __sub_cc(r[threadIdx.x],a[threadIdx.x],b[threadIdx.x]);
  __subcy(cy[threadIdx.x]);
}

/* Compute Rmod <- A + B */ 
/* Input: 0 <= A, B < 3*N */ 
/* Ouput: 0 <= Rmod < 6*N */ 
__device__ void Cuda_Add_mod
(biguint_t Rmod, dbigint_t cy, const biguint_t A,const biguint_t B)
{
  Cuda_Add(Rmod, cy, A, B);
  Cuda_Fully_Normalize(Rmod, cy); 
}

/* Compute Rmod <- Rmod + B */ 
/* Input: 0 <= Rmod, B < 3*N */ 
/* (except when it follows Cuda_Mulint_mod, 0 <= Rmod < 3*N, 0 < B < 7*N ) */ 
/* Ouput: 0 <= Rmod < 6*N */ 
/* (except when it follows Cuda_Mulint_mod, 0 <= Rmod < 10*N) */ 
__device__ void Cuda_Add_mod
(biguint_t Rmod, dbigint_t cy, const biguint_t A)
{
  Cuda_Add(Rmod, cy, Rmod, A);
  Cuda_Fully_Normalize(Rmod, cy);
}

/* Compute Rmod <- Rmod - B */ 
/* Input: 0 <= Rmod, B < 3*N */ 
/* Ouput: 0 <= Rmod < 6*N */ 
__device__ void Cuda_Sub_mod 
(biguint_t Rmod, dbigint_t cy, const biguint_t A)
{
  Cuda_Add (Rmod, cy, Rmod, d_3Ncst);
  Cuda_Subc (Rmod, cy, Rmod, A);
  Cuda_Fully_Normalize(Rmod, cy); 
}

/* Perform one step of REDC */ 
__device__ void Cuda_Mulmod_step
(dbiguint_t r, dbigint_t cy, unsigned int a, unsigned int b)
{
  digit_t h,l;
  int tmp;
  __mul(h,l,a,b);
  __add_cc(r[threadIdx.x],r[threadIdx.x],l);
  __addc_cc(r[threadIdx.x+1],r[threadIdx.x+1],h);
  __addcy2(cy[threadIdx.x+1]);


  __mul(h, l, d_invNcst*r[0], d_Ncst[threadIdx.x]);
  __add_cc(r[threadIdx.x],r[threadIdx.x],l);
  __addc_cc(r[threadIdx.x+1],r[threadIdx.x+1],h);
  __addcy2(cy[threadIdx.x+1]);
 
  //make one round of normalize + a right shift
  __add_cc(r[threadIdx.x],r[threadIdx.x+1],cy[threadIdx.x]);
  tmp=(threadIdx.x==NB_DIGITS-1)?cy[threadIdx.x+1]:0;
  __asm__("addc.s32 %0,%1, 0;" :"=r"(cy[threadIdx.x]): "r"(tmp)); 

  if (threadIdx.x==0)
  {
    cy[NB_DIGITS]=0;
    r[NB_DIGITS]=0;
  }
}

/* Compute r <- 2*a */ 
/* Input: 0 <= a < 3*N */ 
/* Ouput: 0 <= r < 3*N */ 
__device__ void Cuda_Dbl_mod
(biguint_t r, dbigint_t cy, biguint_t a)
{
  __add_cc(r[threadIdx.x],a[threadIdx.x],a[threadIdx.x]);
  __addcy2(r[(threadIdx.x+1)%NB_DIGITS]);

  if (Cuda_Cmp (r, d_3Ncst) >= 0) 
  {
    Cuda_Sub (r, cy, r, d_3Ncst); 
    Cuda_Fully_Normalize(r, cy);  
  }
}


/* Compute r <- A*b */ 
/* Input: 0 < b < 2^SIZE_DIGIT, 0 <= A < 6*N */ 
/* Ouput: 0 <= r < 7*N */ 
__device__ void Cuda_Mulint_mod
(dbiguint_t r, dbigint_t cy, biguint_t A, unsigned int b)
{
  digit_t h,l;
  __mul(h, r[threadIdx.x], A[threadIdx.x], b);
  __add_cc(r[threadIdx.x+1], r[threadIdx.x+1], h);
  __addcy(cy[threadIdx.x+1]);

  //h*2^32+l =A[i]*B[threadIDx.x]
  __mul(h, l, d_invNcst*r[0], d_Ncst[threadIdx.x]);
  __add_cc(r[threadIdx.x], r[threadIdx.x],l);
  __addc_cc(r[threadIdx.x+1], r[threadIdx.x+1],h);
  __addcy2(cy[threadIdx.x+1]);

  __add_cc(r[threadIdx.x], r[threadIdx.x+1], cy[threadIdx.x]);
  __addcy(cy[threadIdx.x]);
  if (threadIdx.x==0)
  {
    cy[NB_DIGITS]=0;
    r[NB_DIGITS]=0;
  }
  
  Cuda_Fully_Normalize(r,cy); 
}

/* Compute r <- A*B */ 
/* Input: 0 <= A, B < 6*N */
/* (except when it follows Cuda_Mulint_mod, 0 <= A < 6*N, 0 < B < 10*N ) */ 
/* Ouput: 0 <= r < 3*N */ 
__device__ void Cuda_Mul_mod 
(biguint_t mul, dbigint_t cy, const biguint_t A, const biguint_t B, dbiguint_t r)
{

  int i;
  digit_t temp=A[threadIdx.x];

  r[threadIdx.x]=0;
  
  for (i=0;i<NB_DIGITS;i++)
    Cuda_Mulmod_step(r,cy,temp,B[i]);

  
  Cuda_Fully_Normalize(r,cy);
  mul[threadIdx.x]=r[threadIdx.x];
}

__device__ void 
Cuda_Square_mod (biguint_t mul, dbigint_t cy, const biguint_t A, dbiguint_t r)
{
  Cuda_Mul_mod(mul,cy,A,A,r);
}

/* 
  Compute silmutaneously:
  (xarg : zarg ) <- [2](xarg : zarg) 
  (xarg2 : zarg2 ) <- (xarg : zarg) + (xarg2 : zarg2) 
*/
__global__ void 
Cuda_Ell_DblAdd (biguint_t *xarg, biguint_t *zarg, biguint_t *x2arg, 
                                       biguint_t *z2arg, unsigned int firstinvd)
{
  __shared__ VOL digit_t b_temp_r[CURVES_BY_BLOCK][NB_DIGITS+1];
  __shared__ VOL carry_t b_cy[CURVES_BY_BLOCK][NB_DIGITS+1]; 

  __shared__ VOL digit_t b_t[CURVES_BY_BLOCK][NB_DIGITS];
  __shared__ VOL digit_t b_u[CURVES_BY_BLOCK][NB_DIGITS];
  __shared__ VOL digit_t b_v[CURVES_BY_BLOCK][NB_DIGITS];
  __shared__ VOL digit_t b_w[CURVES_BY_BLOCK][NB_DIGITS];
  
  volatile unsigned int idx1=blockIdx.x*blockDim.y+threadIdx.y;
  //volatile unsigned int t1=threadIdx.x+1;
  //volatile unsigned int t2=threadIdx.x+NB_DIGITS;
  
  VOL digit_t *t=b_t[threadIdx.y];
  VOL digit_t *u=b_u[threadIdx.y];
  VOL digit_t *v=b_v[threadIdx.y];
  VOL digit_t *w=b_w[threadIdx.y];
  VOL digit_t *temp_r=b_temp_r[threadIdx.y];
  VOL carry_t *cy=b_cy[threadIdx.y];

  //init
  b_cy[threadIdx.y][threadIdx.x]=0; 
  if (threadIdx.x==0)
    b_cy[threadIdx.y][NB_DIGITS]=0; 

  v[threadIdx.x]=x2arg[idx1][threadIdx.x];
  w[threadIdx.x]=z2arg[idx1][threadIdx.x];
  temp_r[threadIdx.x]=zarg[idx1][threadIdx.x];
  u[threadIdx.x]=xarg[idx1][threadIdx.x];

  Cuda_Add_mod(t, cy, v, w);      /* C=x2+z2 */
  Cuda_Sub_mod(v, cy, w);         /* D=x2-z2 */
  Cuda_Add_mod(w, cy, u, temp_r); /* A=x+z */
  Cuda_Sub_mod(u, cy, temp_r);    /* B=x-z */

  Cuda_Mul_mod(t, cy, t, u, temp_r); /* CB=C*B=(xq+zq)(xp-zp) */
  Cuda_Mul_mod(v, cy, v, w, temp_r); /* DA=D*A=(xq-zq)(xp+zp) */

  Cuda_Square_mod(w, cy, w, temp_r); /* AA=A^2 */
  Cuda_Square_mod(u, cy, u, temp_r); /* BB=B^2 */

  Cuda_Mul_mod(temp_r, cy, u, w, temp_r); /* x2=AA*BB */
  xarg[idx1][threadIdx.x]=temp_r[threadIdx.x];

  Cuda_Sub_mod (w, cy, u); /* C= AA-BB */
  Cuda_Mulint_mod (temp_r, cy, w, idx1 + firstinvd); /* d*C */ 
  Cuda_Add_mod (u, cy, temp_r); /* BB+d*C */
 
  Cuda_Mul_mod (w, cy, w, u,temp_r); /* z2=C*(BB+d*C) */
  zarg[idx1][threadIdx.x]=w[threadIdx.x];
 
  Cuda_Add_mod(w, cy, v, t); /* DA+CB mod N */
  Cuda_Sub_mod(v, cy, t); /* DA-CB mod N */

  Cuda_Square_mod(w, cy, w, temp_r); /* (DA+CB)^2 mod N */
  Cuda_Square_mod(v, cy, v, temp_r); /* (DA-CB)^2 mod N */

  /* z0=1 so there is nothing to compute for z0*(DA+CB)^2 */
  Cuda_Dbl_mod(u, cy, v); /* x0=2 x0*(DA-CB)^2 */
  
  x2arg[idx1][threadIdx.x]=w[threadIdx.x];
  z2arg[idx1][threadIdx.x]=u[threadIdx.x];
}

